#include "hip/hip_runtime.h"
// purpose of this file is to house GPU kernels associated with training and evaluating the neural net
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cuda/std/cmath>
#include <hipblas.h>

#include "NeuralNet.h"

//big idea for a neural net: 
// input will be greyscale values for every single pixel in the 4k image
// output will be rgbrgbrgb for each pixel in order -> to easily convert to cimg and to easily grab the error

// idea is 4 layers (input layer, 2 hidden layer, output layer with 3 output neurons) for now with 100 neurons and ReLu activation function layer into the Sigmoid activation function layer which feeds into output
// we will try 3 layers with only sigmoid now, and if that isnt enough we will add a relu layer before and see if that helps

// evaluating inputs for every neuron in a layer and setting the second layer output
// this is accomplished with matrix multiplication

// we will use cuBLAS NVIDIA api for fast matrix multiplication 
void layerMultiplicationWrapper(double* weights, double* inputs, double* biases, double* output, int numNeuronsNextLayer, int numNeuronsCurrentLayer) {
	//wrapping multiplication with cublas	
	double* deviceWeights, * deviceInputs, * deviceBiases;
	hipMalloc(&deviceWeights, sizeof(double) * numNeuronsNextLayer * numNeuronsCurrentLayer);
	hipMalloc(&deviceInputs, sizeof(double) * numNeuronsCurrentLayer);	
	hipMalloc(&deviceBiases, sizeof(double) * numNeuronsNextLayer);
	
	// copying host memory to device
	hipMemcpy(deviceWeights, weights, sizeof(double) * numNeuronsCurrentLayer*numNeuronsNextLayer, hipMemcpyHostToDevice);
	hipMemcpy(deviceInputs, inputs, sizeof(double) * numNeuronsCurrentLayer, hipMemcpyHostToDevice);
	hipMemcpy(deviceBiases, biases, sizeof(double) * numNeuronsNextLayer, hipMemcpyHostToDevice);

	//calling cublas matrix multiply and adding biases vector (this does deviceWeights*deviceInputs + biasVector) and stores the result in the bias vector
	hipblasDgemm(HIPBLAS_OP_N, HIPBLAS_OP_N, numNeuronsNextLayer, 1, numNeuronsCurrentLayer, 1, deviceWeights, numNeuronsNextLayer, deviceInputs, numNeuronsCurrentLayer, 1, deviceBiases, numNeuronsNextLayer);

	// copying result of multiplication and addition back to output host memory
	hipMemcpy(output, deviceBiases, sizeof(double) * numNeuronsNextLayer, hipMemcpyDeviceToHost);

	//freeing device memory
	hipFree(deviceWeights);
	hipFree(deviceInputs);
	hipFree(deviceBiases);
}

// will need to add biases to matrix results if any -> we have as many biases as results
__global__ void biasAdd(double* results,double* biases, int numBiases) {
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < numBiases; i += gridDim.x * blockDim.x) {
		results[i] += biases[i];
	}
}

void biasAddWrapper(double* results, double* biases, int numBiases) {
	double* deviceResults, * deviceBiases;
	hipMalloc(&deviceResults, sizeof(double) * numBiases);
	hipMalloc(&deviceBiases, sizeof(double) * numBiases);

	hipMemcpy(deviceResults, results, sizeof(double) * numBiases, hipMemcpyHostToDevice);
	hipMemcpy(deviceBiases, biases, sizeof(double) * numBiases, hipMemcpyHostToDevice);

	// calling kernel
	biasAdd << <200, 256 >> > (deviceResults, deviceBiases, numBiases);
	
	// copying output
	hipMemcpy(results, deviceResults, sizeof(double) * numBiases, hipMemcpyDeviceToHost);

	// freeing GPU memory
	hipFree(deviceResults);
	hipFree(deviceBiases);
}

__global__ void trainingHelper(net* toTrain, double** currLayerOutput, double** nextDerivatives, double learningRate) {
	int tidX = blockIdx.x * blockDim.x + threadIdx.x;
	int tidY = blockIdx.y * blockDim.y + threadIdx.y;
	for (int i = toTrain->numLayers-1; i >=0; i--) {
		layer* toConsider = toTrain->neuralLayers[i];
		for (int j = tidX; j < toConsider->numNeuronsNextLayer; j += blockDim.x * gridDim.x) {
			for (int z = tidY; z < toConsider->numNeuronsCurrentLayer; z += blockDim.y * gridDim.y) {
				toConsider->weightAdjustments[(j * toConsider->numNeuronsCurrentLayer) + z] = nextDerivatives[i][j] * currLayerOutput[i][j] * (1 - currLayerOutput[i][j]) * toConsider->neuronInputs[z];
				if (i != 0) {
					atomicAdd(&(nextDerivatives[i - 1][z]) , nextDerivatives[i][j] * currLayerOutput[i][j] * (1 - currLayerOutput[i][j]) * toConsider->weightMatrix[(j * toConsider->numNeuronsCurrentLayer) + z]);
				}
			}
			//adjusting the bias while we can
			toConsider->biases[j] -= learningRate * nextDerivatives[i][j] * currLayerOutput[i][j] * (1 - currLayerOutput[i][j]);
		}
	}
	__syncthreads();

	// after syncing threads doing adjustments for all the weights
	for (int i = toTrain->numLayers-1; i >=0; i--) {
		layer* toConsider = toTrain->neuralLayers[i];
		for (int j = tidX; j < toConsider->numNeuronsNextLayer; j += blockDim.x * gridDim.x) {
			for (int z = tidY; z < toConsider->numNeuronsCurrentLayer; z += blockDim.y * gridDim.y) {
				// doing adjustment
				toConsider->weightMatrix[(j * toConsider->numNeuronsCurrentLayer) + z] -= learningRate * toConsider->weightAdjustments[(j * toConsider->numNeuronsCurrentLayer)];
			}
		}
	}

	// now all the weights are changed, we will memcpy the weights in the helper

}

void trainingHelperWrapper(net* toTrain, double* netOutput, double learningRate) {
	net* deviceNet;
	double* deviceNetOutput;
	// allocating and copying the struct to the gpu

	// calling the kernel

	//copying only the updated weights back to the CPU struct

	// freeing memory
}

// applies relu activation function to results
__global__ void reluResults(double* inputs, int numInputs) {
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < numInputs; i += gridDim.x * blockDim.x) {
		inputs[i] = fmaxf(0, inputs[i]);
	}
}

// applies sigmoid activation function to results
__global__ void sigmoidResults(double* inputs, int numInputs) {
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < numInputs; i += gridDim.x * blockDim.x) {
		inputs[i] = 1 / (1 + exp(-inputs[i]));
	}
}

void sigmoidWrapper(double* inputs, int numInputs) {
	double* deviceInputs;
	hipMalloc(&deviceInputs, sizeof(double) * numInputs);

	hipMemcpy(deviceInputs, inputs, sizeof(double) * numInputs, hipMemcpyHostToDevice);

	sigmoidResults << <200, 256 >> > (deviceInputs, numInputs);
	//copying back to host and freeing memory
	hipMemcpy(inputs, deviceInputs, sizeof(double) * numInputs, hipMemcpyDeviceToHost);
	hipFree(deviceInputs);
}

