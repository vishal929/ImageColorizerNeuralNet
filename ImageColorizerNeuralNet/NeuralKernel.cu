#include "hip/hip_runtime.h"
// purpose of this file is to house GPU kernels associated with training and evaluating the neural net
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cuda/std/cmath>
#include <hipblas.h>

#include "NeuralNet.h"
#include "cudaErrorHandler.cuh"

//big idea for a neural net: 
// input will be greyscale values for every single pixel in the 4k image
// output will be rgbrgbrgb for each pixel in order -> to easily convert to cimg and to easily grab the error

// idea is 4 layers (input layer, 2 hidden layer, output layer with 3 output neurons) for now with 100 neurons and ReLu activation function layer into the Sigmoid activation function layer which feeds into output
// we will try 3 layers with only sigmoid now, and if that isnt enough we will add a relu layer before and see if that helps

// evaluating inputs for every neuron in a layer and setting the second layer output
// this is accomplished with matrix multiplication


__global__ void matrixTranspose(double* input, double* output, int inputRows, int inputColumns) {
	int id = blockDim.x * blockIdx.x + threadIdx.x;
	for (int i = id;i < inputRows * inputColumns;i += gridDim.x * blockDim.x) {
		int row = i/inputRows;
		int column = i-(row*inputColumns);
		output[(column * inputRows) + row] = input[(row * inputColumns) + column];
	}
}

// we will use cuBLAS NVIDIA api for fast matrix multiplication 
void layerMultiplicationWrapper(double* weights, double* inputs, double* biases, double* output, int numNeuronsNextLayer, int numNeuronsCurrentLayer) {
	//wrapping multiplication with cublas	
	double* deviceWeights, * deviceInputs, * deviceBiases;
	hipblasHandle_t handle;
	cudaErrorCheck(hipMalloc(&deviceWeights, sizeof(double) * numNeuronsNextLayer * numNeuronsCurrentLayer));
	cudaErrorCheck(hipMalloc(&deviceInputs, sizeof(double) * numNeuronsCurrentLayer));	
	cudaErrorCheck(hipMalloc(&deviceBiases, sizeof(double) * numNeuronsNextLayer));

	int m = 1;
	int k = numNeuronsCurrentLayer;
	int n = numNeuronsNextLayer;
	double identityScalar = 1.0;

	//initializing cublas handle and setting matrices
	hipblasStatus_t status;
	hipblasCreate(&handle);
	status = hipblasSetMatrix(m,k, sizeof(double),inputs,m,deviceInputs,m);
	if (status != HIPBLAS_STATUS_SUCCESS) {
		printf("error with setting matrix 1\n");
	}
	status =hipblasSetMatrix(k,n, sizeof(double), weights, k, deviceWeights,k);
	if (status != HIPBLAS_STATUS_SUCCESS) {
		printf("error with setting matrix 2\n");
	}
	status =hipblasSetMatrix(m,n, sizeof(double), biases, m, deviceBiases, m);
	if (status != HIPBLAS_STATUS_SUCCESS) {
		printf("error with setting matrix 3\n");
	}
	//calling cublas matrix multiply and adding biases vector (this does deviceWeights*deviceInputs + biasVector) and stores the result in the bias vector

	
	status =hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m,n,k,&identityScalar,deviceInputs,m,deviceWeights,k,&identityScalar,deviceBiases,m);

	if (status != HIPBLAS_STATUS_SUCCESS) {
		printf("error with cublas matrix multiplication\n");
	}
	

	// copying result of multiplication and addition back to output host memory
	status=hipblasGetMatrix(m, n, sizeof(double), deviceBiases, m, output, m);
	if (status != HIPBLAS_STATUS_SUCCESS) {
		printf("error with cublas get matrix\n");
	}
	// hipMemcpy(output, deviceBiases, sizeof(double) * numNeuronsNextLayer, hipMemcpyDeviceToHost);

	//destroying handle
	hipblasDestroy(handle);

	hipError_t lastError = hipGetLastError();
	if (lastError != hipSuccess) {
		printf("error with layer multiplication wrapper %s\n", hipGetErrorString(lastError));
	}

	//freeing device memory
	cudaErrorCheck(hipFree(deviceWeights));
	cudaErrorCheck(hipFree(deviceInputs));
	cudaErrorCheck(hipFree(deviceBiases));
}

// will need to add biases to matrix results if any -> we have as many biases as results
__global__ void biasAdd(double* results,double* biases, int numBiases) {
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < numBiases; i += gridDim.x * blockDim.x) {
		results[i] += biases[i];
	}
}

void biasAddWrapper(double* results, double* biases, int numBiases) {
	double* deviceResults, * deviceBiases;
	hipMalloc(&deviceResults, sizeof(double) * numBiases);
	hipMalloc(&deviceBiases, sizeof(double) * numBiases);

	hipMemcpy(deviceResults, results, sizeof(double) * numBiases, hipMemcpyHostToDevice);
	hipMemcpy(deviceBiases, biases, sizeof(double) * numBiases, hipMemcpyHostToDevice);

	// calling kernel
	biasAdd << <200, 256 >> > (deviceResults, deviceBiases, numBiases);
	
	// copying output
	hipMemcpy(results, deviceResults, sizeof(double) * numBiases, hipMemcpyDeviceToHost);

	// freeing GPU memory
	hipFree(deviceResults);
	hipFree(deviceBiases);
}

__global__ void trainingHelper(net* toTrain, double** currLayerOutput, double** nextDerivatives, double learningRate) {
	int tidX = blockIdx.x * blockDim.x + threadIdx.x;
	int tidY = blockIdx.y * blockDim.y + threadIdx.y;
	for (int i = toTrain->numLayers-1; i >=0; i--) {
		layer* toConsider = toTrain->neuralLayers[i];
		for (int j = tidX; j < toConsider->numNeuronsNextLayer; j += blockDim.x * gridDim.x) {
			for (int z = tidY; z < toConsider->numNeuronsCurrentLayer; z += blockDim.y * gridDim.y) {
				toConsider->weightAdjustments[(j * toConsider->numNeuronsCurrentLayer) + z] = nextDerivatives[i][j] * currLayerOutput[i][j] * (1 - currLayerOutput[i][j]) * toConsider->neuronInputs[z];
				if (i != 0) {
					atomicAdd(&(nextDerivatives[i - 1][z]) , nextDerivatives[i][j] * currLayerOutput[i][j] * (1 - currLayerOutput[i][j]) * toConsider->weightMatrix[(j * toConsider->numNeuronsCurrentLayer) + z]);
				}
			}
			//adjusting the bias while we can
			if (tidX == 0 && tidY == 0) {
				toConsider->biases[j] -= learningRate * nextDerivatives[i][j] * currLayerOutput[i][j] * (1 - currLayerOutput[i][j]);
			}
		}
		//syncing threads before moving onto the next layer of backpropogation
		__syncthreads();
	}
	__syncthreads();

	// after syncing threads doing adjustments for all the weights
	for (int i = toTrain->numLayers-1; i >=0; i--) {
		layer* toConsider = toTrain->neuralLayers[i];
		for (int j = tidX; j < toConsider->numNeuronsNextLayer; j += blockDim.x * gridDim.x) {
			for (int z = tidY; z < toConsider->numNeuronsCurrentLayer; z += blockDim.y * gridDim.y) {
				// doing adjustment
				toConsider->weightMatrix[(j * toConsider->numNeuronsCurrentLayer) + z] -= learningRate * toConsider->weightAdjustments[(j * toConsider->numNeuronsCurrentLayer)];
			}
		}
	}

	// now all the weights are changed, we will memcpy the weights in the helper

}

void trainingHelperWrapper(net* toTrain, double* netOutput, double actualR, double actualG, double actualB, double learningRate) {
	double derivatives[3];
	double dEdR = -(((double)(actualR / 255)) - netOutput[0]);
	double dEdG = -(((double)(actualG / 255)) - netOutput[1]);
	double dEdB = -(((double)(actualB / 255)) - netOutput[2]);
	derivatives[0] = dEdR;
	derivatives[1] = dEdG;
	derivatives[2] = dEdB;
	// end of getting initial partial derivatives
	net* deviceNet;
	double **currLayerOutput, ** nextDerivatives;
	// allocating and copying the struct to the gpu
	hipMalloc(&deviceNet, sizeof(net));
	hipMemcpy(deviceNet, toTrain, sizeof(net), hipMemcpyHostToDevice);
	// allocating inner fields of the neural net
	//hipMalloc(&(deviceNet->inputs), sizeof(double) * toTrain->numInputs);
	//hipMemcpy(deviceNet->inputs, toTrain->inputs, sizeof(double) * toTrain->numInputs, hipMemcpyHostToDevice);

	//allocating stored outputs and learning rates we will need
	hipMalloc(&(currLayerOutput), sizeof(double*) * toTrain->numLayers);
	hipMalloc(&(nextDerivatives), sizeof(double*) * toTrain->numLayers);
	// allocating layers
	hipMalloc(&(deviceNet->neuralLayers), sizeof(layer*) * toTrain->numLayers);
	for (int i = 0; i < toTrain->numLayers; i++) {
		hipMalloc(&(deviceNet->neuralLayers[i]), sizeof(layer));
		hipMemcpy(deviceNet->neuralLayers[i], toTrain->neuralLayers[i], sizeof(layer), hipMemcpyHostToDevice);
		// copying inputs, outputs, biases, and allocating adjustments
		hipMalloc(&(deviceNet->neuralLayers[i]->neuronInputs), sizeof(double) * toTrain->neuralLayers[i]->numNeuronsCurrentLayer);
		hipMemcpy(deviceNet->neuralLayers[i]->neuronInputs, toTrain->neuralLayers[i]->neuronInputs, sizeof(double) * toTrain->neuralLayers[i]->numNeuronsCurrentLayer, hipMemcpyHostToDevice);
		hipMalloc(&(deviceNet->neuralLayers[i]->weightMatrix), sizeof(double) * toTrain->neuralLayers[i]->numNeuronsCurrentLayer * toTrain->neuralLayers[i]->numNeuronsNextLayer);
		hipMemcpy(deviceNet->neuralLayers[i]->weightMatrix, toTrain->neuralLayers[i]->weightMatrix, sizeof(double) * toTrain->neuralLayers[i]->numNeuronsCurrentLayer * toTrain->neuralLayers[i]->numNeuronsNextLayer, hipMemcpyHostToDevice);
		hipMalloc(&(deviceNet->neuralLayers[i]->biases), sizeof(double) * toTrain->neuralLayers[i]->numNeuronsNextLayer);
		hipMemcpy(deviceNet->neuralLayers[i]->biases, toTrain->neuralLayers[i]->biases, sizeof(double) * toTrain->neuralLayers[i]->numNeuronsNextLayer, hipMemcpyHostToDevice);
		// allocating adjustments for gpu to fill
		hipMalloc(&(deviceNet->neuralLayers[i]->weightAdjustments), sizeof(double) * toTrain->neuralLayers[i]->numNeuronsCurrentLayer * toTrain->neuralLayers[i]->numNeuronsNextLayer);
		// allocating special memory for training
		hipMalloc(&(currLayerOutput[i]), sizeof(double) * toTrain->neuralLayers[i]->numNeuronsNextLayer);
		hipMalloc(&(nextDerivatives[i]), sizeof(double) * toTrain->neuralLayers[i]->numNeuronsNextLayer);
		if (i==(toTrain->numLayers)-1){
			hipMemcpy(currLayerOutput[i], netOutput, sizeof(double) * 3, hipMemcpyHostToDevice);
			hipMemcpy(nextDerivatives[i], derivatives, sizeof(double) * 3, hipMemcpyHostToDevice);
		} else {
			hipMemcpy(currLayerOutput[i], toTrain->neuralLayers[i + 1]->neuronInputs, sizeof(double) * toTrain->neuralLayers[i + 1]->numNeuronsCurrentLayer, hipMemcpyHostToDevice);
			hipMemset(nextDerivatives[i], 0, sizeof(double) * toTrain->neuralLayers[i]->numNeuronsNextLayer);
		}
	}

	// calling the kernel


	trainingHelper << <200, 256 >> > (deviceNet, currLayerOutput, nextDerivatives, learningRate);

	// freeing memory and copying the updated weights back to the CPU struct -> do not need anything else 
	for (int i = 0; i < toTrain->numLayers; i++) {
		// copying updated weights back to cpu struct and then freeing inner objects
		hipMemcpy(deviceNet->neuralLayers[i]->weightMatrix, toTrain->neuralLayers[i]->weightMatrix, sizeof(double) * toTrain->neuralLayers[i]->numNeuronsCurrentLayer * toTrain->neuralLayers[i]->numNeuronsNextLayer, hipMemcpyDeviceToHost);

		hipFree(nextDerivatives[i]);
		hipFree(currLayerOutput[i]);

		hipFree(deviceNet->neuralLayers[i]->weightAdjustments);
		hipFree(deviceNet->neuralLayers[i]->biases);
		hipFree(deviceNet->neuralLayers[i]->weightMatrix);
		hipFree(deviceNet->neuralLayers[i]->neuronInputs);

		hipFree(deviceNet->neuralLayers[i]);
	}
	
	hipFree(nextDerivatives);
	hipFree(currLayerOutput);

	
	// freeing outer net
	hipFree(deviceNet->neuralLayers);
	hipFree(deviceNet);
}

// applies relu activation function to results
__global__ void reluResults(double* inputs, int numInputs) {
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < numInputs; i += gridDim.x * blockDim.x) {
		inputs[i] = fmaxf(0, inputs[i]);
	}
}

// applies sigmoid activation function to results
__global__ void sigmoidResults(double* inputs, int numInputs) {
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < numInputs; i += gridDim.x * blockDim.x) {
		inputs[i] = 1.0 / (1.0 + exp(-(inputs[i])));
		//output[i] = 90.5;
	}
	
}

void sigmoidWrapper(double* inputs, int numInputs) {
	double* deviceInputs;
	cudaErrorCheck(hipMalloc(&deviceInputs, sizeof(double) * numInputs));
	cudaErrorCheck(hipMemcpy(deviceInputs, inputs, sizeof(double) * numInputs, hipMemcpyHostToDevice));
	sigmoidResults << <200, 256 >> > (deviceInputs , numInputs);
	hipError_t lastError = hipGetLastError();
	if (lastError != hipSuccess) {
		printf( "error with sigmoid wrapper %s\n",hipGetErrorString(lastError));
	}
	//copying back to host and freeing memory
	cudaErrorCheck(hipMemcpy(inputs, deviceInputs, sizeof(double) * numInputs, hipMemcpyDeviceToHost));
	hipFree(deviceInputs);
}

