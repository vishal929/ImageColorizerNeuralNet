#include "hip/hip_runtime.h"
// purpose of this file is to house GPU kernels associated with training and evaluating the neural net
#include "hip/hip_runtime.h"
#include ""
#include "cudaErrorHandler.cuh"
#include "hipblas.h"
#include <vector>
#include <Cimg.h>
#include <string>
#include <iostream>
#include <random>
#include <hiprand.h>
#include "GPUSecondNeuralNet.cuh"
#include "ImageKernel.cuh"



using namespace cimg_library;
using namespace std;



// my idea here is to create a more memory hitting neural net in order to flex the gpu more and speedup training/evaluating and output times
// idea is to take a 1000 x 1000 portion of the image, and guess the middle 500x500 pixel values so the output will be of size 500 x 500 x 3 for RGB values
	// this way, we can extend smaller images with black values to apply to the model and we can batch portions of larger images more easily
	// I will test training and if I cannot get a fit (which is likely with only 100 neurons a layer, I will try increasing the number of neurons)
	// if there is still sufficient gpu memory not being utilized, I can increase the input size and the output size and the number of neurons per layer to get a good mix

CImg<int> getRandomTestImage() {
	string searchName = "./TestData/*";
	WIN32_FIND_DATA FindFileData;
	HANDLE hFind;
	int numPictures = 0;
	hFind = FindFirstFile(searchName.c_str(), &FindFileData);
	if (hFind == INVALID_HANDLE_VALUE) {
		cout << "OH NO Test DATA NOT FOUND!\n";
		//returning empty image
		FindClose(hFind);
		return CImg<int>();
	}
	else {
		numPictures++;
	}

	while (FindNextFile(hFind, &FindFileData)) {
		if ((FindFileData.dwFileAttributes & FILE_ATTRIBUTE_DIRECTORY) == 0) {
			numPictures++;
		}
	}

	if (numPictures == 0) {
		cout << "OH NO!!!!! We have an empty test set! testing aborted\n";
		//returning empty image
		FindClose(hFind);
		return CImg<int>();
	}
	FindClose(hFind);
	// idea is to choose a random picture in the training data folder
	// getting random number in a range
	random_device rando;
	mt19937 gen(rando());
	uniform_int_distribution<> distr(1, numPictures);
	int imageToPick = distr(gen);

	// choosing a random picture
	WIN32_FIND_DATA RandomFileData;
	HANDLE hFindRandom = NULL;

	int currCount = 0;
	string dirName = "TestData/";
	while (currCount != imageToPick) {
		if (hFindRandom == NULL) {
			hFindRandom = FindFirstFile("./TestData/*\0", &RandomFileData);
		}
		else {
			FindNextFile(hFindRandom, &RandomFileData);
		}
		if ((RandomFileData.dwFileAttributes & FILE_ATTRIBUTE_DIRECTORY) == 0) {
			currCount++;
		}
	}
	// now the data is pointed to the picture to choose

	// using cimg to get data for the given picture
	for (int i = 0;i < strlen(RandomFileData.cFileName);i++) {
		dirName.push_back(RandomFileData.cFileName[i]);
	}
	CImg<int> colorPicture(dirName.c_str());
	cout << "grabbed test image: " << dirName << "\n";
	FindClose(hFindRandom);
	return colorPicture;
}

CImg<int> getRandomTrainingImage() {
	string searchName = "./TrainingData/*";
	WIN32_FIND_DATA FindFileData;
	HANDLE hFind;
	int numPictures = 0;
	hFind = FindFirstFile(searchName.c_str(), &FindFileData);
	if (hFind == INVALID_HANDLE_VALUE) {
		cout << "OH NO TRAINING DATA NOT FOUND!\n";
		//returning empty image
		FindClose(hFind);
		return CImg<int>();
	}
	else {
		numPictures++;
	}

	while (FindNextFile(hFind, &FindFileData)) {
		if ((FindFileData.dwFileAttributes & FILE_ATTRIBUTE_DIRECTORY) == 0) {
			numPictures++;
		}
	}

	if (numPictures == 0) {
		cout << "OH NO!!!!! We have an empty training set! Training aborted\n";
		//returning empty image
		FindClose(hFind);
		return CImg<int>();
	}
	FindClose(hFind);
	// idea is to choose a random picture in the training data folder
	// getting random number in a range
	random_device rando;
	mt19937 gen(rando());
	uniform_int_distribution<> distr(1, numPictures);
	int imageToPick = distr(gen);

	// choosing a random picture
	WIN32_FIND_DATA RandomFileData;
	HANDLE hFindRandom = NULL;

	int currCount = 0;
	string dirName = "TrainingData/";
	while (currCount != imageToPick) {
		if (hFindRandom == NULL) {
			hFindRandom = FindFirstFile("./TrainingData/*\0", &RandomFileData);
		}
		else {
			FindNextFile(hFindRandom, &RandomFileData);
		}
		if ((RandomFileData.dwFileAttributes & FILE_ATTRIBUTE_DIRECTORY) == 0) {
			currCount++;
		}
	}
	// now the data is pointed to the picture to choose

	// using cimg to get data for the given picture
	for (int i = 0;i < strlen(RandomFileData.cFileName);i++) {
		dirName.push_back(RandomFileData.cFileName[i]);
	}
	CImg<int> colorPicture(dirName.c_str());
	cout << "grabbed training image: " << dirName << "\n";
	FindClose(hFindRandom);
	return colorPicture;
}

__device__ double sigmoidInput(double input) {
	return 1 / (1 + exp(-input));
}

__global__ void sigmoidMatrix(double* output, int dim) {
	int tidx = blockDim.x * blockIdx.x + threadIdx.x;
	for (int i = tidx;i < dim;i += gridDim.x * blockDim.x) {
		output[i] = sigmoidInput(output[i]);
	}
}

__global__ void reluMatrix(double* output, int dim) {
	int tidx = blockDim.x * blockIdx.x + threadIdx.x;
	for (int i = tidx;i < dim;i += gridDim.x * blockDim.x) {
		output[i] = fmax(0.0, output[i]);
	}
}

//asserting that sigmoid matrix works (we will verify gpu output with cpu)
void sigmoidMatrixTest(double* input, double* output, int dim) {
	//creating copy of input to run sigmoidMatrix on
	double* deviceOutput;
	cudaErrorCheck(hipMalloc(&deviceOutput, sizeof(double) * dim));
	cudaErrorCheck(hipMemcpy(deviceOutput, input, sizeof(double) * dim, hipMemcpyHostToDevice));
	sigmoidMatrix << <20, 512 >> > (deviceOutput, dim);
	cudaErrorCheck(hipMemcpy(output, deviceOutput, sizeof(double) * dim, hipMemcpyDeviceToHost));
	cudaErrorCheck(hipFree(deviceOutput));
	//asserting that kernel ran successfully
	for (int i = 0;i < dim;i++) {
		double cpuSigmoid = 1 / (1 + exp(-input[i]));
		if ( cpuSigmoid != output[i]) {
			cout << "ERROR GPU RETURNED: " << output[i] << "BUT CPU RETURNED: " << cpuSigmoid << "\n";
			return;
		}
		if (isnan(output[i])) {
			cout << "NAN ERROR on INDEX: " << i << "\n";
			return;
		}
	}
	cout << "SIGMOID KERNEL WORKS!\n";
}

void cpuSigmoidMatrix(double* output, int dim) {
	for (int i = 0;i < dim;i++) {
		output[i] = 1 / (1 + exp(-(output[i])));
	}
}

// given weight matrices and biases , we can initialize weights with random small numbers and biases with very small static values
__global__ void initializeGPUNet(double* weights, double* randomNumbers, double* biases, int numInputs, int numOutputs) {
	int tidx = blockDim.x * blockIdx.x + threadIdx.x;
	int tidy = blockDim.y * blockIdx.y + threadIdx.y;
	for (int j = tidx; j < numInputs;j += gridDim.x * blockDim.x) {
		for (int z = tidy; z < numOutputs;z += gridDim.y * blockDim.y) {
			weights[(z * numInputs) + j] = randomNumbers[(z * numInputs) + j]/(numInputs);
			//weights[(z * numInputs) + j] = 0;
			if (tidx == 0) {
				biases[z] = 0;
			}
		}
	}
}

// cpu code for initializing in case gpu code doesnt work out
void initializeWeightsBiases(double* weights, double* biases, int numInputs, int numOutputs) {

}

// reading weights into gpu and returning the gpu allocated struct, we can read layer by layer to save on host memory
// if a particular layer is not defined, we will initialize it with random weights and 0 biases
GPUNet* loadGPUNet() {
	GPUNet* hostStruct = (GPUNet*)malloc(sizeof(GPUNet));
	int numLayers = standardNetSize;
	hostStruct->numLayers = numLayers;
	// allocating outer memory on cpu (inner memory will be allocated on gpu)
	double** hostWeights, **hostBiases, **hostWeightAdjustments, **hostLayerInput, ** hostDeltas;
	int* numInputs, * numOutputs;
	hostWeights = (double**) malloc(sizeof(double*)*numLayers);
	hostBiases = (double**) malloc(sizeof(double*)*numLayers);
	//hostWeightAdjustments = (double**) malloc(sizeof(double*)*numLayers);
	hostDeltas = (double**)malloc(sizeof(double*) * numLayers);
	hostLayerInput= (double**) malloc(sizeof(double*)*numLayers);
	numInputs = (int*)malloc(sizeof(int) * numLayers);
	numOutputs = (int*)malloc(sizeof(int) * numLayers);

	//setting allocating memory to struct
	hostStruct->weights = hostWeights;
	hostStruct->biases = hostBiases;
	//hostStruct->weightAdjustments = hostWeightAdjustments;
	hostStruct->deltas = hostDeltas;
	hostStruct->layerInput = hostLayerInput;
	hostStruct->numInputs = numInputs;
	hostStruct->numOutputs = numOutputs;
	
	string weightName = string("weights.txt");
	// dimension for 2D kernel launches
	dim3 blockShape(32, 32);
	dim3 gridShape(16, 16);
	for (int i = 0;i < numLayers;i++) {
		//adjusting the weight name
		weightName.insert(weightName.begin(), '0' + i);
		// setting our input and output sizes
		int specificInputSize;
		int specificOutputSize;
		if (i == 0) {
			specificInputSize = inputSize;
			specificOutputSize = hiddenLayerNumNeurons;
		}
		else if (i == numLayers - 1) {
			specificInputSize = hiddenLayerNumNeurons;
			specificOutputSize = outputSize;
		}
		else {
			// then this is hidden layer to hidden layer
			specificInputSize = hiddenLayerNumNeurons;
			specificOutputSize = hiddenLayerNumNeurons;
		}
		//setting layer size
		numInputs[i] = specificInputSize;
		numOutputs[i] = specificOutputSize;
		
		// allocating inner memory on gpu
		double* innerDeviceWeights, *deviceInnerBiases, *deviceInnerWeightAdjustments, *deviceInnerLayerInput, *deviceInnerDeltas;
		cudaErrorCheck(hipMalloc(&innerDeviceWeights, sizeof(double) * specificInputSize * specificOutputSize));
		//cudaErrorCheck(hipMalloc(&deviceInnerWeightAdjustments, sizeof(double) * specificInputSize * specificOutputSize));
		cudaErrorCheck(hipMalloc(&deviceInnerDeltas, sizeof(double)  * specificOutputSize));
		cudaErrorCheck(hipMalloc(&deviceInnerBiases, sizeof(double) * specificOutputSize));
		cudaErrorCheck(hipMalloc(&deviceInnerLayerInput, sizeof(double) * specificInputSize * numInputSquares));
		// seeing if we have a weights file for this
		// if not, then we can call the initialize kernel here
		FILE* weightsFile = fopen(weightName.c_str(), "r");
		if (weightsFile == NULL) {
			// then weights dont exist
			// getting random buffer for the allocation kernel
			hiprandGenerator_t gen;
			double* randomBuffer;
			cudaErrorCheck(hipMalloc(&randomBuffer, sizeof(double) * specificInputSize * specificOutputSize));
			// seeding generator
			hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
			hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL));
			// getting random numbers
			hiprandGenerateUniformDouble(gen, randomBuffer, specificInputSize * specificOutputSize);
			// we will call the initialize kernel
			initializeGPUNet <<<gridShape, blockShape >>>(innerDeviceWeights, randomBuffer, deviceInnerBiases, specificInputSize, specificOutputSize);
			// freeing memory and destorying hiprand generator
			cudaErrorCheck(hipFree(randomBuffer));
			hiprandDestroyGenerator(gen);
		}
		else {
			// then we will read in the weights and biases from a file and then copy them to the gpu
			double* hostWeights, * hostBiases;
			hostWeights = (double*)malloc(sizeof(double) * specificInputSize * specificOutputSize);
			hostBiases = (double*)malloc(sizeof(double) * specificOutputSize);
			//reading the weights
			for (int k = 0;k < specificOutputSize;k++) {
				for (int j = 0;j < specificInputSize;j++) {
					if (j == specificInputSize-1) {
						int count = 0;
						do {
							count =fscanf(weightsFile, "%lf\n", &(hostWeights[(k * specificInputSize) + j]));
						} while (count == 0);
					}
					else {
						int count = 0;
						do {
							count = fscanf(weightsFile, "%lf ", &(hostWeights[(k * specificInputSize) + j]));
						} while (count == 0);
					}
				}
			}
			//reading in the biases 
			for (int k = 0;k < specificOutputSize;k++) {
				int count = 0;
				do {
					count =fscanf(weightsFile, "%lf\n", &(hostBiases[k]));
				} while (count == 0);
			}
			//copying the weights and biases to gpu and freeing host memory
			cudaErrorCheck(hipMemcpy(innerDeviceWeights, hostWeights, sizeof(double) * specificInputSize * specificOutputSize, hipMemcpyHostToDevice));
			cudaErrorCheck(hipMemcpy(deviceInnerBiases, hostBiases, sizeof(double) *  specificOutputSize, hipMemcpyHostToDevice));
			free(hostWeights);
			free(hostBiases);
			cout << "loaded existing weights for layer: " << i << " \n";

		}
		// closing the filestream and resetting the string
		if (weightsFile != NULL) fclose(weightsFile);
		weightName = string("weights.txt");

		// setting the inner pointers to the gpu allocated memory
		hostWeights[i] = innerDeviceWeights;
		hostBiases[i] = deviceInnerBiases;
		//hostWeightAdjustments[i] = deviceInnerWeightAdjustments;
		hostLayerInput[i] = deviceInnerLayerInput;
		hostDeltas[i] = deviceInnerDeltas;
	}

	// returning the struct containing gpu allocated pointer
	return hostStruct;
}


// writing weights from gpu to filesystem, we can write layer by layer to save on host memory
void writeGPUNet(GPUNet* net) {
	string weightFile = string("weights.txt");
	for (int i = 0;i < net->numLayers;i++) {
		//setting up file to be written to (overwritten)
		weightFile.insert(weightFile.begin(), '0' + i);
		remove(weightFile.c_str());
		//ofstream weightText;
		//weightText.open(weightFile, ios::trunc | ios::out);
		FILE* toWrite = fopen(weightFile.c_str(), "w");
		// setting up host memory as intermediate for writing to file
		double* hostWeights, * hostBiases;
		hostWeights = (double*)malloc(sizeof(double) * net->numInputs[i] * net->numOutputs[i]);
		hostBiases = (double*)malloc(sizeof(double) * net->numOutputs[i]);
		// copying the weights and biases from gpu memory to host memory
		cudaErrorCheck(hipMemcpy(hostWeights, net->weights[i], sizeof(double) * net->numInputs[i] * net->numOutputs[i], hipMemcpyDeviceToHost));
		cudaErrorCheck(hipMemcpy(hostBiases, net->biases[i], sizeof(double) *  net->numOutputs[i], hipMemcpyDeviceToHost));

		// writing with fprint
		//writing weights first
		for (int j = 0;j < net->numOutputs[i];j++) {
			for (int z = 0;z < net->numInputs[i]; z++) {
				if (z == net->numInputs[i] - 1) {
					int count = 0;
					do {
						count = fprintf(toWrite, "%.15lf\n", hostWeights[(j * net->numInputs[i]) + z]);
					} while (count == 0);
				}
				else {
					int count = 0;
					do {
						count = fprintf(toWrite, "%.15lf ", hostWeights[(j * net->numInputs[i]) + z]);
					} while (count == 0);
				}
			}
		}

		//writing biases
		for (int j = 0;j < net->numOutputs[i];j++) {
			int count = 0;
			do { 
				count = fprintf(toWrite, "%.15lf\n", hostBiases[j]);
			} while (count == 0);
		}

		// freeing host memory,closing file, and resetting weight file name
		free(hostWeights);
		free(hostBiases);
		fclose(toWrite);
		weightFile = string("weights.txt");
	}
}

// keep in mind we might have multiple inputs, so we can take the average gradient towards the end
// this kernel is agnostic to the number of input patches to the neural net
__global__ void backPropogationGradientCalculationOutputLayer(double* actualOutputs, double* netOutputs, double* outputDeltas, int numOutputs) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int i = tid;i < numOutputs; i += gridDim.x * blockDim.x) {
		//sigmoid derivative
		for (int j = 0;j < numInputSquares;j++) {
			
			outputDeltas[i] += (netOutputs[(i*numInputSquares)+j]-actualOutputs[(i*numInputSquares)+j]) * netOutputs[(i*numInputSquares)+j] * (1 - netOutputs[(i*numInputSquares)+j]);
		}
		//relu derivative
		/*
		if (netOutputs[i] == 0) {
			
			outputDeltas[i] = (netOutputs[i]-actualOutputs[i]) * 0;
		}
		else {
			outputDeltas[i] = (netOutputs[i]-actualOutputs[i]) * 1;
		}
		*/
	}
}

__global__ void backPropogationGradientCalculationHiddenLayer(double* layerOutput, double* outputDeltas, double* nextLayerWeights, double* nextLayerDeltas, int numLayerOutputs, int numNextLayerOutputs) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	//firstly getting sum delta of associated successive layer, and then multiplying derivative as well
	for (int i = tid;i < numLayerOutputs; i += gridDim.x * blockDim.x) {
		for (int j = 0;j < numNextLayerOutputs;j++) {
			//delta calculation
			//sigmoid derivative
			outputDeltas[i] += nextLayerDeltas[j] * nextLayerWeights[(j * numLayerOutputs) + i] * layerOutput[i] * (1-layerOutput[i]);
			//relu derivative
			/*
			if (layerOutput[i] == 0) {
					outputDeltas[i] += nextLayerDeltas[j] * nextLayerWeights[(j * numLayerOutputs) + i] * 0;
			}
			else {
					outputDeltas[i] += nextLayerDeltas[j] * nextLayerWeights[(j * numLayerOutputs) + i] * 1;
			} */
			
			
		}
	}
}

//function for getting average deltas for the updates
// num outputs is the output size of the net itself on a single input
// num inputs is the number of input patches given to the net
__global__ void averageBatchGradient(double* gradient, int numInputs, int numOutputs) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int i = tid;i < numOutputs;i+=gridDim.x*blockDim.x) {
		// average calculation
		gradient[i] /= numInputs;
	}
}

//function that actually updates the weights based on the deltas, we will just use the last input as the basis for backtracking
__global__ void finalizeUpdate(double* layerInputs, double* layerDeltas, double* layerBiases, double learningRate, double* weights, int numOutputs, int numInputs) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int i = tid;i < numOutputs;i+=gridDim.x*blockDim.x) {
		for (int j = 0;j < numInputs;j++) {
		
			// just arbitrarily picking the first input to use for backprop updating
			weights[(i * numInputs) + j] -= learningRate * layerDeltas[i] * layerInputs[(j*numInputSquares)];
		}
		layerBiases[i] -= learningRate * layerDeltas[i];
	}
}

void batchBackPropogation(GPUNet* toTrain,double* finalOutput,double* netOutput,double learningRate) {
	double* deviceFinalOutput, *deviceNetOutput;
	cudaErrorCheck(hipMalloc(&deviceFinalOutput, sizeof(double) * outputSize * numInputSquares));
	cudaErrorCheck(hipMalloc(&deviceNetOutput, sizeof(double) * outputSize * numInputSquares));
	cudaErrorCheck(hipMemcpy(deviceNetOutput, netOutput, sizeof(double) * outputSize * numInputSquares, hipMemcpyHostToDevice));
	cudaErrorCheck(hipMemcpy(deviceFinalOutput, finalOutput, sizeof(double) * outputSize *numInputSquares , hipMemcpyHostToDevice));
	for (int i = toTrain->numLayers - 1;i >= 0;i--) {
		//memsetting the delta for this layer to all zeroes
		cudaErrorCheck(hipMemset(toTrain->deltas[i], 0, sizeof(double) * toTrain->numOutputs[i] ));
		if (i == toTrain->numLayers - 1) {
			// special kernel for output layer
			backPropogationGradientCalculationOutputLayer << <20, 256>> > (deviceFinalOutput,deviceNetOutput,toTrain->deltas[i],toTrain->numOutputs[i]);
			averageBatchGradient << <20, 256 >> > (toTrain->deltas[i], numInputSquares, toTrain->numOutputs[i]);
			// freeing device memory we no longer need
			cudaErrorCheck(hipFree(deviceFinalOutput));
			cudaErrorCheck(hipFree(deviceNetOutput));
		}
		else {
			// then we have a hidden layer
			backPropogationGradientCalculationHiddenLayer <<<20,256>>> (toTrain->layerInput[i+1],toTrain->deltas[i],toTrain->weights[i+1],toTrain->deltas[i+1],toTrain->numOutputs[i],toTrain->numOutputs[i+1]);
		}
	}

	// doing weight updates
	for (int i = 0;i < toTrain->numLayers;i++) {
		finalizeUpdate << <20, 256>> > (toTrain->layerInput[i], toTrain->deltas[i], toTrain->biases[i],learningRate, toTrain->weights[i], toTrain->numOutputs[i], toTrain->numInputs[i]);
	}
	// now weights are updated
}

// does the initial step of backpropogation while evaluating (sets the adjustments to be the derivative values) (we are using sigmoid, so the derivative is just output(1-output)
__global__ void backPropogateGPUInputHelper(double* weightAdjustment, double* outputs, int numInputs, int numOutputs) {
	int tidx = blockDim.x * blockIdx.x + threadIdx.x;
	int tidy = blockDim.y * blockIdx.y + threadIdx.y;
	for (int i = tidx;i < numOutputs; i += blockDim.x * gridDim.x) {
		for (int j = tidy;j < numInputs;j += blockDim.y * gridDim.y) {
			weightAdjustment[(i * numInputs) + j] = outputs[i] * (1-outputs[i]);
			
		}
	}
}

//verifying that my cublas logic and cpu matrix logic are the same -> returns false if not the same
boolean testCublas() {
	// test weights is a 5x6 matrix with values from 1 to 30
	// test biases is a vector of output size with values 1 to 5
	// test inputs is a vector of inputs with values from  1 to 6
	int numInputs = 6;
	int numOutputs = 5;
	double* testWeights, * testBias, * testInput;
	testWeights = (double*)malloc(sizeof(double) * numInputs * numOutputs);
	testBias = (double*)malloc(sizeof(double) * numOutputs);
	testInput = (double*)malloc(sizeof(double) * numInputs);
	for (int i = 0;i < numInputs * numOutputs;i++) {
		if (i < 5) {
			testBias[i] = i + 1;
		}

		if (i < 6) {
			testInput[i] = i + 1;
		}

		testWeights[i] = i + 1;
	}
	// cuda logic
	double* deviceCPUTestOutput = (double*)malloc(sizeof(double) * numOutputs);
	double* deviceWeights, * deviceBiases, * deviceOutput, * deviceInput;
	cudaErrorCheck(hipMalloc(&deviceWeights, sizeof(double) * numInputs * numOutputs));
	cudaErrorCheck(hipMalloc(&deviceBiases, sizeof(double) *  numOutputs));
	cudaErrorCheck(hipMalloc(&deviceOutput, sizeof(double) *  numOutputs));
	cudaErrorCheck(hipMalloc(&deviceInput, sizeof(double) *  numInputs));
	
	cudaErrorCheck(hipMemcpy(deviceWeights, testWeights, sizeof(double) * numInputs * numOutputs, hipMemcpyHostToDevice));
	cudaErrorCheck(hipMemcpy(deviceBiases, testBias, sizeof(double) *  numOutputs, hipMemcpyHostToDevice));
	cudaErrorCheck(hipMemcpy(deviceInput, testInput, sizeof(double) *  numInputs, hipMemcpyHostToDevice));

	// copying biases to output buffer
	cudaErrorCheck(hipMemcpy(deviceOutput, deviceBiases, sizeof(double) *  numOutputs, hipMemcpyDeviceToDevice));

	hipblasHandle_t handle;
	hipblasStatus_t status;
	hipblasCreate(&handle);
		
	int m = 1;
	int k = numInputs;
	int n = numOutputs;
	double identityScalar = 1.0;

	//calling cublas matrix multiply and adding biases vector (this does deviceWeights*deviceInputs + biasVector) and stores the result in the layerOutput vector

	status = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &identityScalar, deviceInput, m, deviceWeights, k, &identityScalar, deviceOutput, m);


	if (status != HIPBLAS_STATUS_SUCCESS) {
		printf("error with cublas matrix multiplication\n");
	}

	cudaErrorCheck(hipMemcpy(deviceCPUTestOutput, deviceOutput, sizeof(double) * numOutputs, hipMemcpyDeviceToHost));

	cudaErrorCheck(hipFree(deviceWeights));
	cudaErrorCheck(hipFree(deviceBiases));
	cudaErrorCheck(hipFree(deviceOutput));
	cudaErrorCheck(hipFree(deviceInput));	

	//destroying handle
	hipblasDestroy(handle);

	// doing cpu matrix multiplication and addition
	double* result = (double*)malloc(sizeof(double) * numOutputs);

	for (int k = 0;k < numOutputs;k++) {
		double sum = 0;
		for (int i = 0;i < numInputs;i++) {
			sum += testWeights[(k * numInputs) + i] * testInput[i];
		}
		result[k] = sum + testBias[k];
	}


	//verification
	for (int i = 0;i < numOutputs;i++) {
		if (deviceCPUTestOutput[i] != result[i]) {
			cout << "VERY BIG ISSUE!\n";
			return false;
		}
	}

	free(result);
	free(testWeights);
	free(testInput);
	free(testBias);

	return true;

}

// evaluating the entire gpu net with cublas and some input
void evaluateGPUNet(GPUNet* toEvaluate, double* inputs, double* outputBuffer) {
	cudaErrorCheck(hipMemcpy(toEvaluate->layerInput[0], inputs, sizeof(double) * toEvaluate->numInputs[0], hipMemcpyHostToDevice));
	// going through every layer and applying cublas
	//wrapping multiplication with cublas	
	/*
	int* numInputs = (int*) malloc(sizeof(int) * toEvaluate->numLayers);
	int* numOutputs = (int*)malloc(sizeof(int) * toEvaluate->numLayers);
	cudaErrorCheck(hipMemcpy(numInputs, toEvaluate->numInputs, sizeof(int) * toEvaluate->numLayers, hipMemcpyDeviceToHost));
	cudaErrorCheck(hipMemcpy(numOutputs, toEvaluate->numOutputs, sizeof(int) * toEvaluate->numLayers, hipMemcpyDeviceToHost));
	*/

	//layer input and output to keep track of 
	double* layerOutput;

	// sizes for 2d kernels
	dim3 blockShape(32, 32);
	dim3 gridShape(16, 16);
	
	//initializing cublas handle and setting matrices
	hipblasHandle_t handle;
	hipblasStatus_t status;
	hipblasCreate(&handle);
	// looping for gpu multiplication and addition of layers
	for (int i = 0;i < toEvaluate->numLayers;i++) {
		
		int m = 1;
		int k = toEvaluate->numInputs[i];
		int n = toEvaluate->numOutputs[i];
		double identityScalar = 1.0;

		

		//allocating buffer for the input
		/*
		cudaErrorCheck(hipMalloc((&layerInput), sizeof(double) * toEvaluate->numInputs[i]));
		if (i == 0) {
			cudaErrorCheck(hipMemcpy(layerInput, inputs, sizeof(double) * toEvaluate->numOutputs[0], hipMemcpyHostToDevice));
		}
		else {
			cudaErrorCheck(hipMemcpy(layerInput, layerOutput, sizeof(double) * toEvaluate->numInputs[i], hipMemcpyDeviceToDevice));
			cudaErrorCheck(hipFree(layerOutput));
		} */

		//setting up output as a copy of biases
		cudaErrorCheck(hipMalloc(&layerOutput, sizeof(double) * toEvaluate->numOutputs[i]));
		cudaErrorCheck(hipMemcpy(layerOutput, toEvaluate->biases[i], sizeof(double) * toEvaluate->numOutputs[i], hipMemcpyDeviceToDevice));
		
		//calling cublas matrix multiply and adding biases vector (this does deviceWeights*deviceInputs + biasVector) and stores the result in the layerOutput vector

		status = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &identityScalar, toEvaluate->layerInput[i], m, toEvaluate->weights[i], k, &identityScalar, layerOutput, m);

		if (status != HIPBLAS_STATUS_SUCCESS) {
			printf("error with cublas matrix multiplication\n");
		}

		// applying sigmoid to the output layer and relu to the other layers
	
		if (i == toEvaluate->numLayers - 1) {
			sigmoidMatrix <<<20,512 >>> (layerOutput, toEvaluate->numOutputs[i]);
		}
		else {
			
			reluMatrix<<<20,512 >>> (layerOutput, toEvaluate->numOutputs[i]);
		}
		
		cudaErrorCheck(hipGetLastError());

		//double* sigmoidedCheck = (double*)malloc(sizeof(double) * toEvaluate->numInputs[i] * toEvaluate->numOutputs[i]);
		//cudaErrorCheck(hipMemcpy(sigmoidedCheck,layerOutput,sizeof(double)*toEvaluate->numOutputs[i]))
		
		/*
		double* toSigmoid = (double*)malloc(sizeof(double) * toEvaluate->numOutputs[i] );
		hipMemcpy(toSigmoid, layerOutput, sizeof(double) * toEvaluate->numOutputs[i], hipMemcpyDeviceToHost);
		cpuSigmoidMatrix(toSigmoid, toEvaluate->numOutputs[i]);
		hipMemcpy(layerOutput, toSigmoid, sizeof(double) * toEvaluate->numOutputs[i], hipMemcpyHostToDevice);
		free(toSigmoid);
		*/

		//freeing device memory
		if (i == toEvaluate->numLayers - 1) {
			//copying output to final buffer
			cudaErrorCheck(hipMemcpy(outputBuffer, layerOutput, sizeof(double) * toEvaluate->numOutputs[i], hipMemcpyDeviceToHost));
		}
		else {
			//copying output to next layers input
			cudaErrorCheck(hipMemcpy(toEvaluate->layerInput[i + 1], layerOutput, sizeof(double) * toEvaluate->numOutputs[i], hipMemcpyDeviceToDevice));
		}
		//doing the propogation helper step
		//backPropogateGPUInputHelper <<<gridShape, blockShape>>> (toEvaluate->weightAdjustments[i], layerOutput, toEvaluate->numInputs[i], toEvaluate->numOutputs[i]);
		cudaErrorCheck(hipGetLastError());
		cudaErrorCheck(hipFree(layerOutput));

	}
	//destroying handle
	hipblasDestroy(handle);
}

// we can set the input to the be batched (i.e we keep the weights matrix, but the inputs will be a matrix instead of a vector, so we can calculate multiple outputs at the same time) 
// so, if we have a 1920x1080 image, our inputs will be a matrix where every column contains the 50x50 patch, and the output matrix will be a (3 x (1920x1080)) matrix for every pixels RGB value
// if this is too much, we can set a max batch input size, and then do a few different matrix multiplications -> max batch size defined in GPUSecondNeuralNet.cuh
void batchedGPUEvaluate(GPUNet* toEvaluate, double* inputs, double* outputBuffer) {
	cudaErrorCheck(hipMemcpy(toEvaluate->layerInput[0], inputs, sizeof(double) * toEvaluate->numInputs[0]*numInputSquares, hipMemcpyHostToDevice));
	// going through every layer and applying cublas
	//wrapping multiplication with cublas	
	//layer input and output to keep track of 
	double* layerOutput;

	// sizes for 2d kernels
	dim3 blockShape(32, 32);
	dim3 gridShape(16, 16);

	//initializing cublas handle and setting matrices
	hipblasHandle_t handle;
	hipblasStatus_t status;
	hipblasCreate(&handle);
	// looping for gpu multiplication and addition of layers
	for (int i = 0;i < toEvaluate->numLayers;i++) {

		int m = numInputSquares;
		int k = toEvaluate->numInputs[i];
		int n = toEvaluate->numOutputs[i];
		double identityScalar = 1.0;



		//allocating buffer for the input
		/*
		cudaErrorCheck(hipMalloc((&layerInput), sizeof(double) * toEvaluate->numInputs[i]));
		if (i == 0) {
			cudaErrorCheck(hipMemcpy(layerInput, inputs, sizeof(double) * toEvaluate->numOutputs[0], hipMemcpyHostToDevice));
		}
		else {
			cudaErrorCheck(hipMemcpy(layerInput, layerOutput, sizeof(double) * toEvaluate->numInputs[i], hipMemcpyDeviceToDevice));
			cudaErrorCheck(hipFree(layerOutput));
		} */

		//setting up output as a copy of biases in every column for our batched output
		cudaErrorCheck(hipMalloc(&layerOutput, sizeof(double) * toEvaluate->numOutputs[i] * numInputSquares));
		double* biases = (double*)malloc(sizeof(double) * toEvaluate->numOutputs[i]);
		cudaErrorCheck(hipMemcpy(biases, toEvaluate->biases[i], sizeof(double) * toEvaluate->numOutputs[i], hipMemcpyDeviceToHost));
		double* biasMatrix = (double*)malloc(sizeof(double) * toEvaluate->numOutputs[i] * numInputSquares);
		for (int z = 0;z < numInputSquares; z++) {
			for (int y = 0;y < toEvaluate->numOutputs[i];y++) {
				biasMatrix[(y * numInputSquares) + z] = biases[y];
			}
		}
		//copying matrix to the cudaMallocedBuffer

		cudaErrorCheck(hipMemcpy(layerOutput, biasMatrix, sizeof(double) * toEvaluate->numOutputs[i] * numInputSquares, hipMemcpyHostToDevice));
		//freeing host memory
		free(biases);
		free(biasMatrix);
		

		//calling cublas matrix multiply and adding biases matrix (this does deviceWeights*deviceInputs + biasVector) and stores the result in the layerOutput matrix

		status = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &identityScalar, toEvaluate->layerInput[i], m, toEvaluate->weights[i], k, &identityScalar, layerOutput, m);

		if (status != HIPBLAS_STATUS_SUCCESS) {
			printf("error with cublas matrix multiplication\n");
		}

		// applying sigmoid to the output

		//sigmoidMatrix << <20, 512 >> > (layerOutput, toEvaluate->numOutputs[i] * numInputSquares);
		// applying sigmoid to the output layer and relu to the other layers
	
		//if (i == toEvaluate->numLayers - 1) {
			sigmoidMatrix <<<20,512 >>> (layerOutput, toEvaluate->numOutputs[i] * numInputSquares);
		//}
		//else {
			
		//	reluMatrix<<<20,512 >>> (layerOutput, toEvaluate->numOutputs[i]*numInputSquares);
		//}
		cudaErrorCheck(hipGetLastError());

		//double* sigmoidedCheck = (double*)malloc(sizeof(double) * toEvaluate->numInputs[i] * toEvaluate->numOutputs[i]);
		//cudaErrorCheck(hipMemcpy(sigmoidedCheck,layerOutput,sizeof(double)*toEvaluate->numOutputs[i]))

		/*
		double* toSigmoid = (double*)malloc(sizeof(double) * toEvaluate->numOutputs[i] );
		hipMemcpy(toSigmoid, layerOutput, sizeof(double) * toEvaluate->numOutputs[i], hipMemcpyDeviceToHost);
		cpuSigmoidMatrix(toSigmoid, toEvaluate->numOutputs[i]);
		hipMemcpy(layerOutput, toSigmoid, sizeof(double) * toEvaluate->numOutputs[i], hipMemcpyHostToDevice);
		free(toSigmoid);
		*/

		//freeing device memory
		if (i == toEvaluate->numLayers - 1) {
			//copying output to final buffer
			cudaErrorCheck(hipMemcpy(outputBuffer, layerOutput, sizeof(double) * toEvaluate->numOutputs[i] * numInputSquares, hipMemcpyDeviceToHost));
		}
		else {
			//copying output to next layers input
			cudaErrorCheck(hipMemcpy(toEvaluate->layerInput[i + 1], layerOutput, sizeof(double) * toEvaluate->numOutputs[i] * numInputSquares, hipMemcpyDeviceToDevice));
		}
		//doing the propogation helper step
		//backPropogateGPUInputHelper << <gridShape, blockShape >> > (toEvaluate->weightAdjustments[i], layerOutput, toEvaluate->numInputs[i], toEvaluate->numOutputs[i]);
		cudaErrorCheck(hipGetLastError());
		cudaErrorCheck(hipFree(layerOutput));

	}
	//destroying handle
	hipblasDestroy(handle);
}



// calculates the adjustments based on the derivatives and sets up derivatives for next layer of backpropogation
__global__ void weightAdjust(double* weightAdjustments,double* biases, double* weights, double* derivatives, double* nextDerivatives, int numInputs, int numOutputs, double learningRate) {
	int tidx = blockDim.x * blockIdx.x + threadIdx.x;
	int tidy = blockDim.y * blockIdx.y + threadIdx.y;
	for (int i = tidx; i < numOutputs;i += gridDim.x * blockDim.x) {
		for (int j = tidy;j < numInputs;j += gridDim.y * blockDim.y) {
			weightAdjustments[(i * numInputs) + j] *= derivatives[i];
			atomicAdd(&(nextDerivatives[j]),weightAdjustments[(i * numInputs) + j] * weights[(i*numInputs)+j] );
		}
		
	}
	__syncthreads();
	for (int i = tidx; i < numOutputs;i += gridDim.x * blockDim.x) {
		if (tidy == 0) {
			biases[i] -= learningRate * weightAdjustments[(i * numInputs)];
		}
	}	
}

//actually adjusts the weights and biases
__global__ void finalizeWeightAdjust(double* weights, double* weightAdjustments, double* inputs, int numInputs, int numOutputs,double learningRate) {
	int tidx = blockIdx.x * blockDim.x + threadIdx.x;
	int tidy = blockIdx.y * blockDim.y + threadIdx.y;
	for (int i = tidx;i < numOutputs;i += gridDim.x * blockDim.x) {
		for (int j = tidy;j < numInputs; j+= gridDim.y*blockDim.y) {
			weights[(i*numInputs) + j] -= learningRate * weightAdjustments[(i*numInputs)+j] * inputs[j];
			/*
			if (inputs[j] == 0) {
				printf("NOOOOOOOOOOOOOOO\n");
			} */
		}
	}
}

// backpropogation of the entire net given the output values and the actual values
void backPropogateGPUNet(GPUNet* toBackProp, double* outputBuffer, double* actualRed, double* actualGreen, double* actualBlue, double learningRate) {
	// going through each layer and setting weight adjustments	
	// then performing the adjustments
	double* derivatives = (double*)malloc(sizeof(double) * outputSize);
	double* nextDerivatives;
	//setting the initial partial derivatives
	for (int i = 0;i < outputSize;i++) {
		if (i < outputSquareSide * outputSquareSide) {
			//red
			derivatives[i]=(-(actualRed[i] - outputBuffer[i]));
		}
		else if (i < 2 * outputSquareSide * outputSquareSide) {
			//green
			derivatives[i]=(-(actualGreen[i-(outputSquareSide*outputSquareSide)] - outputBuffer[i]));
		}
		else {
			//blue
			derivatives[i]=(-(actualBlue[i-(2*outputSquareSide*outputSquareSide)] - outputBuffer[i]));
		}
		/*
		if (i % 3 == 0) {
			//red
			derivatives[i]=(-(actualRed[i/3] - outputBuffer[i]));
		}
		else if (i % 3 == 1) {
			//green
			derivatives[i]=(-(actualGreen[i/3] - outputBuffer[i]));
		}
		else {
			//blue
			derivatives[i]=(-(actualBlue[i/3] - outputBuffer[i]));
		}
		/*
		derivatives[i]=(-(actualRed[i] - outputBuffer[i]));
		derivatives[i+1]=(-(actualGreen[i] - outputBuffer[i+1]));
		derivatives[i+2]=(-(actualBlue[i] - outputBuffer[i+2]));
		*/
	}

	dim3 dimBlock(32, 32);
	dim3 dimGrid;
	dimGrid.x = 16;
	dimGrid.y = 16;

	//copying derivatives to deviceDerivatives pointer
	double* deviceDerivatives;
	hipMalloc(&deviceDerivatives, sizeof(double) * outputSize);
	cudaErrorCheck(hipMemcpy(deviceDerivatives, derivatives, sizeof(double) * outputSize, hipMemcpyHostToDevice));

	for (int z = toBackProp->numLayers-1;z >= 0;z--) {
		cudaErrorCheck(hipMalloc(&nextDerivatives, sizeof(double) * toBackProp->numInputs[z]));
		weightAdjust<<<dimGrid,dimBlock>>>(toBackProp->weightAdjustments[z], toBackProp->biases[z], toBackProp->weights[z], deviceDerivatives,nextDerivatives, toBackProp->numInputs[z], toBackProp->numOutputs[z], learningRate);
		//freeing memory and setting up for next iteration
		cudaErrorCheck(hipFree(deviceDerivatives));
		deviceDerivatives = nextDerivatives;
	}

	cudaErrorCheck(hipFree(deviceDerivatives));

	//actually doing the weight adjustments
	for (int z = 0;z < toBackProp->numLayers;z++) {	
		finalizeWeightAdjust<<<dimGrid,dimBlock>>>(toBackProp->weights[z], toBackProp->weightAdjustments[z], toBackProp->layerInput[z],toBackProp->numInputs[z] ,toBackProp->numOutputs[z], learningRate);
	}


	// freeing host memory
	free(derivatives);
	
}

// given an image, we will run the net on it and output the result image, will also fill an error buffer
void outputFromGPUNet(char* imageName, char* outputImageName) {
	GPUNet* toTrain= loadGPUNet();
	double* layer2Weights = (double*)malloc(sizeof(double) * toTrain->numInputs[1] * toTrain->numOutputs[1]);
	hipMemcpy(layer2Weights, toTrain->weights[1], sizeof(double) * toTrain->numInputs[1] * toTrain->numOutputs[1], hipMemcpyDeviceToHost);
	cout << "layer 2 weight 40: " << layer2Weights[40] << " \n";
	free(layer2Weights);
	//will divide the image into squares and output it
	// going through patches, running the net on each patch, and then adding to the final buffer
	int imageCount = 0;
	//getting black and white image
	CImg<int> chosenImage(imageName);
	// converting image to black and white
	int* bwBuffer = chosenImage.data();
	int* finalBuffer = (int*)malloc(sizeof(int) * 3 * chosenImage.height() * chosenImage.width());
	//makeImageBlackAndWhiteWrapper(randomImage.data(), randomImage.data() + (randomImage.height() * randomImage.width()), randomImage.data() + (2 * randomImage.height() * randomImage.width()), bwBuffer, randomImage.height(), randomImage.width());
	// crop parts to fit neural net input size
	// we will crop into perfect squares and then combine them to get the final image (we do not need to combine them for training though)
		for (int i = 0;i < chosenImage.height();i += outputSquareSide) {
			for (int j = 0;j < chosenImage.width();j += outputSquareSide) {
				// getting the square for both the bw image and color image
				int* bwSquare = (int*)malloc(sizeof(int) * squareSide * squareSide);

				getSquareWrapper(bwBuffer, bwSquare, squareSide, chosenImage.height(), chosenImage.width(), i, j);

				
				CImg<int> testColorSquare(bwSquare, outputSquareSide, outputSquareSide);
				testColorSquare.save("outputSquare.jpg", imageCount);


				// scale pixels by 255 for both bw image and color image
				double* scaledBWSquare = (double*)malloc(sizeof(double) * squareSide * squareSide);
				pixelScaleWrapper(bwSquare, scaledBWSquare, squareSide, squareSide, inputPixelScaler);
				free(bwSquare);

				// evaluate net for each part of the image
				// we will have outputSize number of outputs, and we will train the net so that the output[0] is first pixels R, output[1],output[2] represent first pixels G and B value
				double* outputBuffer = (double*)malloc(sizeof(double) * outputSize);

				evaluateGPUNet(toTrain, scaledBWSquare, outputBuffer);

				
				double* copyOfOutput = (double*)malloc(sizeof(double) * outputSize);
				memcpy(copyOfOutput, outputBuffer, sizeof(double) * outputSize);
				for (int k = 0;k < outputSize;k++) {
					copyOfOutput[k] *= inputPixelScaler;
				}
				CImg<double> testBuffer(copyOfOutput, outputSquareSide, outputSquareSide, 1, 3);
				testBuffer.save("OutputSquareImage.jpg", imageCount);
				free(copyOfOutput);
				imageCount++;

			//copying output buffer to finalBuffer positions	
			for (int k = 0;k < outputSquareSide;k++) {
				for (int y = 0;y < outputSquareSide;y++) {
					if (i + k < chosenImage.height() && y + j < chosenImage.width()){
						finalBuffer[((i + k) * chosenImage.width()) + (j + y)] = outputBuffer[(k * outputSquareSide) + y]*inputPixelScaler;
						finalBuffer[(chosenImage.width() * chosenImage.height())+((i + k) * chosenImage.width()) + (j + y)] = outputBuffer[(outputSquareSide*outputSquareSide)+(k * outputSquareSide) + y]*inputPixelScaler;
						finalBuffer[(2*chosenImage.width() * chosenImage.height())+((i + k) * chosenImage.width()) + (j + y)] = outputBuffer[(2*outputSquareSide*outputSquareSide)+(k * outputSquareSide) + y]*inputPixelScaler;
					}
				}
			}	

			// freeing memory we no longer need
			free(scaledBWSquare);
			free(outputBuffer);
			}
		}
		// creating the color image and saving it to disk
		
		CImg<int> newColorImage(finalBuffer,  chosenImage.width(), chosenImage.height(), 1, 3);
		newColorImage.save(outputImageName);
		
}

// finish this method and randomize training
double testFromTestData(GPUNet* toTest) {
	// grabbing a random image
	CImg<int> randomImage = getRandomTestImage();
	// loading the net
	if (toTest== NULL) {
		toTest = loadGPUNet();
	}
	
	
	size_t freeMem;
	size_t totalMem;
	hipMemGetInfo(&freeMem, &totalMem);
	cout << "total memory of gpu: " << totalMem << "\n";
	cout << "total free memory of gpu after loading net: " << freeMem << "\n";
	

	// converting image to black and white
	int* bwBuffer = (int*)malloc(sizeof(int) * randomImage.height() * randomImage.width());
	makeImageBlackAndWhiteWrapper(randomImage.data(), randomImage.data() + (randomImage.height() * randomImage.width()), randomImage.data() + (2 * randomImage.height() * randomImage.width()), bwBuffer, randomImage.height(), randomImage.width());
	int* finalBuffer = (int*)malloc(sizeof(int) * 3* randomImage.height() * randomImage.width());
	// crop parts to fit neural net input size
	double testError = 0;
	int numBatchImages = 0;
	double* bwMatrix = (double*) malloc(sizeof(double) * numInputSquares * inputSize);
	double* colorMatrix = (double*) malloc(sizeof(double)*numInputSquares *outputSize);
	int* rowIndices = (int*)malloc(sizeof(int) * numInputSquares);
	int* colIndices = (int*)malloc(sizeof(int) * numInputSquares);

	cout << "running batch evaluate on all pixels for the image! ...\n";
	for (int i = 0;i < randomImage.height();i += outputSquareSide ) {
		for (int j = 0;j < randomImage.width();j += outputSquareSide ) {
			//cout << "on pixel i: " << i << "and pixel j: " << j << "\n";
			// getting the square for both the bw image and color image
			int* bwSquare = (int*)malloc(sizeof(int) * squareSide * squareSide);
			
			int* redSquare = (int*)malloc(sizeof(int) * outputSquareSide * outputSquareSide);
			int* greenSquare = (int*)malloc(sizeof(int) * outputSquareSide * outputSquareSide);
			int* blueSquare = (int*)malloc(sizeof(int) * outputSquareSide * outputSquareSide);
			

			/*
			hipMemGetInfo(&freeMem, &totalMem);
			cout << "total memory of gpu: " << totalMem << "\n";
			cout << "total free memory of gpu before grabbing color squares: " << freeMem << "\n";
			*/

			// getting squares
			getSquareWrapper(bwBuffer, bwSquare, squareSide, randomImage.height(), randomImage.width(), i, j);
			getSquareWrapper(randomImage.data(), redSquare, outputSquareSide, randomImage.height(), randomImage.width(), i, j);
			getSquareWrapper(randomImage.data() + (randomImage.height() * randomImage.width()), greenSquare, outputSquareSide, randomImage.height(), randomImage.width(), i, j);
			getSquareWrapper(randomImage.data() + (2 * randomImage.height() * randomImage.width()), blueSquare, outputSquareSide, randomImage.height(), randomImage.width(), i, j);
			
			//cout << "got squares!\n";
			/*
			hipMemGetInfo(&freeMem, &totalMem);
			cout << "total memory of gpu: " << totalMem << "\n";
			cout << "total free memory of gpu after grabbing color squares: " << freeMem << "\n";
			*/

			/*
			hipMemGetInfo(&freeMem, &totalMem);
			cout << "total memory of gpu: " << totalMem << "\n";
			cout << "total free memory of gpu before grabbing bw square: " << freeMem << "\n";
			*/


			/*
			hipMemGetInfo(&freeMem, &totalMem);
			cout << "total memory of gpu: " << totalMem << "\n";
			cout << "total free memory of gpu after grabbing bw square: " << freeMem << "\n";
			*/

			// scale pixels by 255 for both bw image and color image
			/*
			hipMemGetInfo(&freeMem, &totalMem);
			cout << "total memory of gpu: " << totalMem << "\n";
			cout << "total free memory of gpu before scaling squares: " << freeMem << "\n";
			*/

			/*
			hipMemGetInfo(&freeMem, &totalMem);
			cout << "total memory of gpu: " << totalMem << "\n";
			cout << "total free memory of gpu after scaling squares: " << freeMem << "\n";
			*/
			double* scaledBWSquare = (double*)malloc(sizeof(double) * squareSide * squareSide);
			pixelScaleWrapper(bwSquare, scaledBWSquare, squareSide, squareSide, inputPixelScaler);
			free(bwSquare);
			double* scaledRedSquare = (double*)malloc(sizeof(double) * outputSquareSide * outputSquareSide);
			pixelScaleWrapper(redSquare, scaledRedSquare, outputSquareSide, outputSquareSide,inputPixelScaler);
			free(redSquare);
			double* scaledGreenSquare = (double*)malloc(sizeof(double) * outputSquareSide * outputSquareSide);
			pixelScaleWrapper(greenSquare, scaledGreenSquare, outputSquareSide, outputSquareSide,inputPixelScaler);
			free(greenSquare);
			double* scaledBlueSquare = (double*)malloc(sizeof(double) * outputSquareSide * outputSquareSide);
			pixelScaleWrapper(blueSquare, scaledBlueSquare, outputSquareSide, outputSquareSide,inputPixelScaler);
			free(blueSquare);

			//cout << "scaled squares!\n";

			// copying memory to the accumulated input buffers so that we can evaluate the entire net on multiple inputs at once
			for (int z = 0;z < inputSize;z++) {
				bwMatrix[(z * numInputSquares) + numBatchImages] = scaledBWSquare[z];
			}

			for (int z = 0;z < outputSquareSide * outputSquareSide;z++) {
				colorMatrix[(z * numInputSquares) + numBatchImages] = scaledRedSquare[z];
				colorMatrix[((z+(outputSquareSide*outputSquareSide)) * numInputSquares) + numBatchImages] = scaledGreenSquare[z];
				colorMatrix[((z+(2*outputSquareSide*outputSquareSide)) * numInputSquares) + numBatchImages] = scaledBlueSquare[z];
			}

			rowIndices[numBatchImages] = i;
			colIndices[numBatchImages] = j;

			// freeing memory we no longer need
			free(scaledBWSquare);
			free(scaledRedSquare);
			free(scaledGreenSquare);
			free(scaledBlueSquare);

			numBatchImages++;

			//cout << "loaded batch Image: " << numBatchImages << "\n";

			if (numBatchImages != numInputSquares && (j+outputSquareSide < randomImage.width() || i+outputSquareSide < randomImage.height())) {
				continue;
			}

			if (numBatchImages!=numInputSquares) {
				//then we need to artificially fill the rest of the inputs, since we hit the end of input
				// and we should make sure that these do not contribute to our error calculations
				for (int z = numBatchImages;z < numInputSquares;z++) {
					for (int y = 0;y < inputSize;y++) {
						bwMatrix[(y * numInputSquares) + z] = 0;
					}
				}
			}
			// otherwise we have filled the input matrix to batch evaluate
			
			

			// evaluate net for each square vector in the input
			double* outputBuffer = (double*)malloc(sizeof(double) * numInputSquares * outputSize );
			/*
			hipMemGetInfo(&freeMem, &totalMem);
			cout << "total free memory of gpu before evaluating net: " << freeMem << "\n";
			*/

			/*
			hipMemGetInfo(&freeMem, &totalMem);
			cout << "total memory of gpu: " << totalMem << "\n";
			cout << "total free memory of gpu before evaluating net: " << freeMem << "\n";
			*/

			batchedGPUEvaluate(toTest, bwMatrix, outputBuffer);
			//cout << "finished batch evaluate for " << numBatchImages << " squares\n";

			//evaluateGPUNet(toTest, scaledBWSquare, outputBuffer);

			// error calculation 
			for (int z = 0;z < outputSquareSide * outputSquareSide;z++) {
				for (int k = 0;k < numInputSquares;k++) {
					if (k < numBatchImages) {
						testError += 0.5 * pow(outputBuffer[(z * numInputSquares) + k] - scaledRedSquare[z], 2);
						testError += 0.5 * pow(outputBuffer[(((outputSquareSide * outputSquareSide) + z) * numInputSquares) + k] - scaledGreenSquare[z], 2);
						testError += 0.5 * pow(outputBuffer[(((2 * outputSquareSide * outputSquareSide) + z) * numInputSquares) + k] - scaledBlueSquare[z], 2);
					}
				}
			}


			/*
			hipMemGetInfo(&freeMem, &totalMem);
			cout << "total memory of gpu: " << totalMem << "\n";
			cout << "total free memory of gpu after evaluating net: " << freeMem << "\n";
			*/

			// iterating through vector output and setting final buffer data
			for (int k = 0;k < numBatchImages;k++) {
				int row = rowIndices[k];
				int col = colIndices[k];
				for (int z = 0;z < outputSquareSide*outputSquareSide;z++) {
					// error adjustment
					int rowAdjust = z / outputSquareSide;
					int colAdjust = z % outputSquareSide;
					int adjustedRow = row + rowAdjust;
					int adjustedCol = col + colAdjust;
					// all the reds are together, greens are together and blues are together
					if (adjustedRow < randomImage.height() && adjustedCol < randomImage.width()) {
						// then we put colors in their correct spot
						finalBuffer[((adjustedRow) * randomImage.width()) + (adjustedCol)] = outputBuffer[(z * numInputSquares)+k]*inputPixelScaler;
						finalBuffer[(randomImage.width() * randomImage.height())+((adjustedRow) * randomImage.width()) + (adjustedCol)] = outputBuffer[(((outputSquareSide*outputSquareSide)+z) * numInputSquares)+k]*inputPixelScaler;
						finalBuffer[(2*randomImage.width() * randomImage.height())+((adjustedRow) * randomImage.width()) + (adjustedCol)] = outputBuffer[(((2*outputSquareSide*outputSquareSide)+z) * numInputSquares)+k]*inputPixelScaler;	
					}
				
				}
			}
			

			/*
			hipMemGetInfo(&freeMem, &totalMem);
			cout << "total memory of gpu: " << totalMem << "\n";
			cout << "total free memory of gpu before backprop net: " << freeMem << "\n";
			*/

			//resetting the number of batch images
			numBatchImages = 0;	
			free(outputBuffer);
		}
	}
	free(bwMatrix);
	free(colorMatrix);
	free(rowIndices);
	free(colIndices);
	

	// printing out the current training error
	printf("obtained test error: %lf\n", testError);

	/*/
	double* layer2Weights = (double*)malloc(sizeof(double) * toTrain->numInputs[1] * toTrain->numOutputs[1]);
	hipMemcpy(layer2Weights, toTrain->weights[1], sizeof(double) * toTrain->numInputs[1] * toTrain->numOutputs[1], hipMemcpyDeviceToHost);
	cout << "layer 2 weight 10: " << layer2Weights[10] << " \n";
	free(layer2Weights);
	*/

	CImg<int> testedOutput(finalBuffer, randomImage.width(), randomImage.height(), 1, 3);
		
	testedOutput.save("testWhileTrainOutput.png");

	//freeing allocated memory
	
	free(bwBuffer);
	free(finalBuffer);

	// returning the double error
	return testError;
}

	
	


// gpu trains the neural net on a random image from the dataset given a learning rate
void trainFromDataSet(double learningRate) {
	// loading the net
	GPUNet* toTrain = loadGPUNet();
	
	size_t freeMem;
	size_t totalMem;
	hipMemGetInfo(&freeMem, &totalMem);
	cout << "total memory of gpu: " << totalMem << "\n";
	cout << "total free memory of gpu after loading net: " << freeMem << "\n";
	

	while (true) {
		
		int trainCount = 0;
		// our loss function is (1/2)SUM(actualColor-predictedColor)^2
		double currTrainingError = (double)0;
		// buffers for batch training
		double* bwMatrix = (double*)malloc(sizeof(double) * inputSize * numInputSquares);
		double* colorMatrix = (double*)malloc(sizeof(double) * numInputSquares * outputSize);
		//int* rowIndices = (int*)malloc(sizeof(int) * inputSize);
		//int* colIndices = (int*)malloc(sizeof(int) * inputSize);
		int currBatches = 0;
		while (trainCount != epochNum*numInputSquares) {
			// pick a random image from the training dataset
			CImg<int> randomImage = getRandomTrainingImage();
			// convert image to black and white
			int* bwBuffer = (int*)malloc(sizeof(int) * randomImage.height() * randomImage.width());
			makeImageBlackAndWhiteWrapper(randomImage.data(), randomImage.data() + (randomImage.height() * randomImage.width()), randomImage.data() + (2 * randomImage.height() * randomImage.width()), bwBuffer, randomImage.height(), randomImage.width());
			// pick a random pixel from the image
			random_device rando;
			mt19937 gen(rando());
			uniform_int_distribution<> row(0, randomImage.height()-1);
			int randomPixelRow = row(gen);

			random_device rando2;
			mt19937 gen2(rando2());
			uniform_int_distribution<> col(0, randomImage.width()-1);
			int randomPixelCol = col(gen);

			// getting squares around the pixel for black and white and color
				int* bwSquare = (int*)malloc(sizeof(int) * squareSide * squareSide);

				int* redSquare = (int*)malloc(sizeof(int) * outputSquareSide * outputSquareSide);
				int* greenSquare = (int*)malloc(sizeof(int) * outputSquareSide * outputSquareSide);
				int* blueSquare = (int*)malloc(sizeof(int) * outputSquareSide * outputSquareSide);

				getSquareWrapper(bwBuffer, bwSquare, squareSide, randomImage.height(), randomImage.width(), randomPixelRow, randomPixelCol);
				getSquareWrapper(randomImage.data(), redSquare, outputSquareSide, randomImage.height(), randomImage.width(), randomPixelRow, randomPixelCol);
				getSquareWrapper(randomImage.data() + (randomImage.height() * randomImage.width()), greenSquare, outputSquareSide, randomImage.height(), randomImage.width(), randomPixelRow, randomPixelCol);
				getSquareWrapper(randomImage.data() + (2 * randomImage.height() * randomImage.width()), blueSquare, outputSquareSide, randomImage.height(), randomImage.width(), randomPixelRow, randomPixelCol);

				//debug
				/*
				for (int z = 0;z < outputSquareSide * outputSquareSide;z++) {
					cout << "red pixel value: " << redSquare[z] << "\n";
				} */


				// scaling squares by 255
				double* scaledBWSquare = (double*)malloc(sizeof(double) * squareSide * squareSide);
				pixelScaleWrapper(bwSquare, scaledBWSquare, squareSide, squareSide, inputPixelScaler);
				free(bwSquare);
				double* scaledRedSquare = (double*)malloc(sizeof(double) * outputSquareSide * outputSquareSide);
				pixelScaleWrapper(redSquare, scaledRedSquare, outputSquareSide, outputSquareSide, inputPixelScaler);
				free(redSquare);
				double* scaledGreenSquare = (double*)malloc(sizeof(double) * outputSquareSide * outputSquareSide);
				pixelScaleWrapper(greenSquare, scaledGreenSquare, outputSquareSide, outputSquareSide, inputPixelScaler);
				free(greenSquare);
				double* scaledBlueSquare = (double*)malloc(sizeof(double) * outputSquareSide * outputSquareSide);
				pixelScaleWrapper(blueSquare, scaledBlueSquare, outputSquareSide, outputSquareSide, inputPixelScaler);
				free(blueSquare);

				// filling the bwmatrix and color matrix for batched training
				for (int z = 0;z < inputSize;z++) {
					bwMatrix[(z * numInputSquares) + currBatches] = scaledBWSquare[z];
				}
				//double* accumulatedActualValues = (double*)malloc(sizeof(double) * outputSize);

				for (int z = 0;z < outputSquareSide * outputSquareSide;z++) {
					/*
					accumulatedActualValues[z] = scaledRedSquare[z];
					accumulatedActualValues[(outputSquareSide * outputSquareSide) + z] = scaledGreenSquare[z];
					accumulatedActualValues[(2*outputSquareSide * outputSquareSide) + z] = scaledBlueSquare[z];
					*/

					colorMatrix[(z * numInputSquares) + currBatches] = scaledRedSquare[z];
					colorMatrix[((z + (outputSquareSide * outputSquareSide)) * numInputSquares) + currBatches] = scaledGreenSquare[z];
					colorMatrix[((z + (2 * outputSquareSide * outputSquareSide)) * numInputSquares) + currBatches] = scaledBlueSquare[z];

				}

				//rowIndices[currBatches] = i;
				//colIndices[currBatches] = j;

				// freeing unnecessary data 

				free(bwBuffer);
				free(scaledBWSquare);
				free(scaledRedSquare);
				free(scaledGreenSquare);
				free(scaledBlueSquare);

				// checking how many batches we have 
				currBatches++;
				trainCount++;
				cout << "FINISHED EPOCH: " << trainCount << "\n";
				if (currBatches != numInputSquares) {
					// then we grab more squares
					continue;
				}

				//resetting curr batches
				currBatches = 0;

				// running output of the scaled black and white squares

				//while (true) {
					double* outputBuffer = (double*)malloc(sizeof(double) * outputSize * numInputSquares);

					int count = 0;
					//evaluateGPUNet(toTrain, scaledBWSquare, outputBuffer);
					batchedGPUEvaluate(toTrain, bwMatrix, outputBuffer);


					// incrementing training error	
					/*
					for (int z = 0;z < outputSize;z++) {
						currTrainingError += 0.5 * pow(outputBuffer[z] - accumulatedActualValues[z], 2);
					} */
					for (int z = 0;z < outputSize * numInputSquares;z++) {
						currTrainingError += 0.5 * pow(outputBuffer[z] - colorMatrix[z], 2);
						//debug
						/*
						cout << "obtained pixel value: " << outputBuffer[z] * inputPixelScaler << "\n";
						cout << "actual pixel value: " << colorMatrix[z] * inputPixelScaler << "\n";
						*/

					}
					//debug
					/*
					cout << "current error: " << currTrainingError << "\n";
					currTrainingError = 0;
					*/


					// backprop for the batch
					batchBackPropogation(toTrain, colorMatrix, outputBuffer, learningRate);

					//debugging for output	
					/*
					for (int z = 0;z < outputSize;z++) {
						outputBuffer[z] *= inputPixelScaler;
					}


					if (count == 0) {
						for (int z = 0;z < outputSize;z++) {
							accumulatedActualValues[z] *= inputPixelScaler;
						}
						CImg<double> actualColor(accumulatedActualValues, outputSquareSide, outputSquareSide, 1, 3);
						actualColor.save("actualPatch.jpg", trainCount);
						for (int z = 0;z < outputSize;z++) {
							accumulatedActualValues[z] /= inputPixelScaler;
						}
						count++;
					}

					CImg<double> guessedColor(outputBuffer, outputSquareSide, outputSquareSide, 1, 3);
					guessedColor.save("guessedPatch.jpg", trainCount);
					*/

					// freeing memory we no longer need
					/*
					free(accumulatedActualValues);
					free(scaledBWSquare);
					free(scaledRedSquare);
					free(scaledGreenSquare);
					free(scaledBlueSquare);
					free(bwBuffer);
					*/
					free(outputBuffer);
				//}


				// printing out the current training error
				printf("current training error: %lf\n", currTrainingError);
		}
		free(bwMatrix);
		free(colorMatrix);
		//free(rowIndices);
		//free(colIndices);
		// then we will perform a test of error on a random test data image
		// writing weights back to filesystem now that the epochLimit was reached
		cout << "writing updated weights to filesystem\n";
		writeGPUNet(toTrain);
		cout << "testing net on testing data now...\n";
		// forget testing for now, since we need to train a lot more
		double testError = testFromTestData(toTrain);
	}	
}

