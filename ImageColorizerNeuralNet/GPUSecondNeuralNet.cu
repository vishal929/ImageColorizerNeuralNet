#include "hip/hip_runtime.h"
// purpose of this file is to house GPU kernels associated with training and evaluating the neural net
#include "hip/hip_runtime.h"
#include ""
#include "cudaErrorHandler.cuh"
#include "hipblas.h"
#include <vector>
#include <Cimg.h>
#include <string>
#include <iostream>
#include <random>
#include <hiprand.h>
#include "GPUSecondNeuralNet.cuh"
#include "ImageKernel.cuh"



using namespace std;
using namespace cimg_library;



// my idea here is to create a more memory hitting neural net in order to flex the gpu more and speedup training/evaluating and output times
// idea is to take a 1000 x 1000 portion of the image, and guess the middle 500x500 pixel values so the output will be of size 500 x 500 x 3 for RGB values
	// this way, we can extend smaller images with black values to apply to the model and we can batch portions of larger images more easily
	// I will test training and if I cannot get a fit (which is likely with only 100 neurons a layer, I will try increasing the number of neurons)
	// if there is still sufficient gpu memory not being utilized, I can increase the input size and the output size and the number of neurons per layer to get a good mix

CImg<int> getRandomTrainingImage() {
	string searchName = "./TrainingData/*";
	WIN32_FIND_DATA FindFileData;
	HANDLE hFind;
	int numPictures = 0;
	hFind = FindFirstFile(searchName.c_str(), &FindFileData);
	if (hFind == INVALID_HANDLE_VALUE) {
		cout << "OH NO TRAINING DATA NOT FOUND!\n";
		//returning empty image
		FindClose(hFind);
		return CImg<int>();
	}
	else {
		numPictures++;
	}

	while (FindNextFile(hFind, &FindFileData)) {
		if ((FindFileData.dwFileAttributes & FILE_ATTRIBUTE_DIRECTORY) == 0) {
			numPictures++;
		}
	}

	if (numPictures == 0) {
		cout << "OH NO!!!!! We have an empty training set! Training aborted\n";
		//returning empty image
		FindClose(hFind);
		return CImg<int>();
	}
	// idea is to choose a random picture in the training data folder
	// getting random number in a range
	random_device rando;
	mt19937 gen(rando());
	uniform_int_distribution<> distr(1, numPictures);
	int imageToPick = distr(gen);

	// choosing a random picture

	int currCount = 0;
	string dirName = "TrainingData/";
	while (currCount != imageToPick) {
		if (hFind == NULL) {
			hFind = FindFirstFile("./TrainingData/*\0", &FindFileData);
		}
		else {
			FindNextFile(hFind, &FindFileData);
		}
		if ((FindFileData.dwFileAttributes & FILE_ATTRIBUTE_DIRECTORY) == 0) {
			currCount++;
		}
	}
	// now the data is pointed to the picture to choose

	// using cimg to get data for the given picture
	for (int i = 0;i < strlen(FindFileData.cFileName);i++) {
		dirName.push_back(FindFileData.cFileName[i]);
	}
	CImg<int> colorPicture(dirName.c_str());
	cout << "grabbed training image: " << dirName << "\n";
	FindClose(hFind);
	return colorPicture;
}

__device__ double sigmoidInput(double input) {
	return 1 / (1 + exp(-input));
}

__global__ void sigmoidMatrix(double* output, int dim) {
	int tidx = blockDim.x * blockIdx.x + threadIdx.x;
	for (int i = tidx;i < dim;i += gridDim.x * blockDim.x) {
		output[i] = sigmoidInput(output[i]);
	}
}

//asserting that sigmoid matrix works (we will verify gpu output with cpu)
void sigmoidMatrixTest(double* input, double* output, int dim) {
	//creating copy of input to run sigmoidMatrix on
	double* deviceOutput;
	cudaErrorCheck(hipMalloc(&deviceOutput, sizeof(double) * dim));
	cudaErrorCheck(hipMemcpy(deviceOutput, input, sizeof(double) * dim, hipMemcpyHostToDevice));
	sigmoidMatrix << <20, 512 >> > (deviceOutput, dim);
	cudaErrorCheck(hipMemcpy(output, deviceOutput, sizeof(double) * dim, hipMemcpyDeviceToHost));
	cudaErrorCheck(hipFree(deviceOutput));
	//asserting that kernel ran successfully
	for (int i = 0;i < dim;i++) {
		double cpuSigmoid = 1 / (1 + exp(-input[i]));
		if ( cpuSigmoid != output[i]) {
			cout << "ERROR GPU RETURNED: " << output[i] << "BUT CPU RETURNED: " << cpuSigmoid << "\n";
			return;
		}
		if (isnan(output[i])) {
			cout << "NAN ERROR on INDEX: " << i << "\n";
			return;
		}
	}
	cout << "SIGMOID KERNEL WORKS!\n";
}

void cpuSigmoidMatrix(double* output, int dim) {
	for (int i = 0;i < dim;i++) {
		output[i] = 1 / (1 + exp(-output[i]));
	}
}

// given weight matrices and biases , we can initialize weights with random small numbers and biases with 0
__global__ void initializeGPUNet(double* weights, double* randomNumbers, double* biases, int numInputs, int numOutputs) {
	int tidx = blockDim.x * blockIdx.x + threadIdx.x;
	int tidy = blockDim.y * blockIdx.y + threadIdx.y;
	for (int j = tidx; j < numInputs;j += gridDim.x * blockDim.x) {
		for (int z = tidy; z < numOutputs;z += gridDim.y * blockDim.y) {
			weights[(z * numInputs) + j] = randomNumbers[(z * numInputs) + j];
			if (tidx == 0) {
				biases[z] = 0;
			}
		}
	}
}

// cpu code for initializing in case gpu code doesnt work out
void initializeWeightsBiases(double* weights, double* biases, int numInputs, int numOutputs) {

}

// reading weights into gpu and returning the gpu allocated struct, we can read layer by layer to save on host memory
// if a particular layer is not defined, we will initialize it with random weights and 0 biases
GPUNet* loadGPUNet() {
	GPUNet* hostStruct = (GPUNet*)malloc(sizeof(GPUNet));
	int numLayers = standardNetSize;
	hostStruct->numLayers = numLayers;
	// allocating outer memory on cpu (inner memory will be allocated on gpu)
	double** hostWeights, **hostBiases, **hostWeightAdjustments, **hostLayerInput;
	int* numInputs, * numOutputs;
	hostWeights = (double**) malloc(sizeof(double*)*numLayers);
	hostBiases = (double**) malloc(sizeof(double*)*numLayers);
	hostWeightAdjustments = (double**) malloc(sizeof(double*)*numLayers);
	hostWeightAdjustments = (double**) malloc(sizeof(double*)*numLayers);
	hostLayerInput= (double**) malloc(sizeof(double*)*numLayers);
	numInputs = (int*)malloc(sizeof(int) * numLayers);
	numOutputs = (int*)malloc(sizeof(int) * numLayers);

	//setting allocating memory to struct
	hostStruct->weights = hostWeights;
	hostStruct->biases = hostBiases;
	hostStruct->weightAdjustments = hostWeightAdjustments;
	hostStruct->layerInput = hostLayerInput;
	hostStruct->numInputs = numInputs;
	hostStruct->numOutputs = numOutputs;
	
	string weightName = string("weights.txt");
	// dimension for 2D kernel launches
	dim3 blockShape(16, 16);
	dim3 gridShape(32, 32);
	for (int i = 0;i < numLayers;i++) {
		//adjusting the weight name
		weightName.insert(weightName.begin(), '0' + i);
		// setting our input and output sizes
		int specificInputSize;
		int specificOutputSize;
		if (i == 0) {
			specificInputSize = inputSize;
			specificOutputSize = hiddenLayerNumNeurons;
		}
		else if (i == numLayers - 1) {
			specificInputSize = hiddenLayerNumNeurons;
			specificOutputSize = outputSize;
		}
		else {
			// then this is hidden layer to hidden layer
			specificInputSize = hiddenLayerNumNeurons;
			specificOutputSize = hiddenLayerNumNeurons;
		}
		//setting layer size
		numInputs[i] = specificInputSize;
		numOutputs[i] = specificOutputSize;
		
		// allocating inner memory on gpu
		double* innerDeviceWeights, *deviceInnerBiases, *deviceInnerWeightAdjustments, *deviceInnerLayerInput;
		cudaErrorCheck(hipMalloc(&innerDeviceWeights, sizeof(double) * specificInputSize * specificOutputSize));
		cudaErrorCheck(hipMalloc(&deviceInnerWeightAdjustments, sizeof(double) * specificInputSize * specificOutputSize));
		cudaErrorCheck(hipMalloc(&deviceInnerBiases, sizeof(double) * specificOutputSize));
		cudaErrorCheck(hipMalloc(&deviceInnerLayerInput, sizeof(double) * specificInputSize));
		// seeing if we have a weights file for this
		// if not, then we can call the initialize kernel here
		FILE* weightsFile = fopen(weightName.c_str(), "r");
		if (weightsFile == NULL) {
			// then weights dont exist
			// getting random buffer for the allocation kernel
			hiprandGenerator_t gen;
			double* randomBuffer;
			cudaErrorCheck(hipMalloc(&randomBuffer, sizeof(double) * specificInputSize * specificOutputSize));
			// seeding generator
			hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
			hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);
			// getting random numbers
			hiprandGenerateUniformDouble(gen, randomBuffer, specificInputSize * specificOutputSize);
			// we will call the initialize kernel
			initializeGPUNet <<<gridShape, blockShape >>>(innerDeviceWeights, randomBuffer, deviceInnerBiases, specificInputSize, specificOutputSize);
			// freeing memory and destorying hiprand generator
			cudaErrorCheck(hipFree(randomBuffer));
			hiprandDestroyGenerator(gen);
		}
		else {
			// then we will read in the weights and biases from a file and then copy them to the gpu
			double* hostWeights, * hostBiases;
			hostWeights = (double*)malloc(sizeof(double) * specificInputSize * specificOutputSize);
			hostBiases = (double*)malloc(sizeof(double) * specificOutputSize);
			//reading the weights
			for (int i = 0;i < specificOutputSize;i++) {
				for (int j = 0;j < specificInputSize;j++) {
					if (j == specificInputSize-1) {
						fscanf(weightsFile, "%lf\n", &(hostWeights[(i * specificInputSize) + j]));
					}
					else {
						fscanf(weightsFile, "%lf ", &(hostWeights[(i * specificInputSize) + j]));
					}
				}
			}
			//reading in the biases 
			for (int i = 0;i < specificOutputSize;i++) {
				fscanf(weightsFile, "%lf\n", &(hostBiases[i]));
			}
			//copying the weights and biases to gpu and freeing host memory
			cudaErrorCheck(hipMemcpy(innerDeviceWeights, hostWeights, sizeof(double) * specificInputSize * specificOutputSize, hipMemcpyHostToDevice));
			cudaErrorCheck(hipMemcpy(deviceInnerBiases, hostBiases, sizeof(double) *  specificOutputSize, hipMemcpyHostToDevice));
			free(hostWeights);
			free(hostBiases);

		}
		// closing the filestream and resetting the string
		if (weightsFile != NULL) fclose(weightsFile);
		weightName = string("weights.txt");

		// setting the inner pointers to the gpu allocated memory
		hostWeights[i] = innerDeviceWeights;
		hostBiases[i] = deviceInnerBiases;
		hostWeightAdjustments[i] = deviceInnerWeightAdjustments;
		hostLayerInput[i] = deviceInnerLayerInput;
	}

	// returning the struct containing gpu allocated pointer
	return hostStruct;
}


// writing weights from gpu to filesystem, we can write layer by layer to save on host memory
void writeGPUNet(GPUNet* net) {
	string weightFile = string("weights.txt");
	for (int i = 0;i < net->numLayers;i++) {
		//setting up file to be written to (overwritten)
		weightFile.insert(weightFile.begin(), '0' + i);
		remove(weightFile.c_str());
		FILE* toWrite = fopen(weightFile.c_str(), "w");
		// setting up host memory as intermediate for writing to file
		double* hostWeights, * hostBiases;
		hostWeights = (double*)malloc(sizeof(double) * net->numInputs[i] * net->numOutputs[i]);
		hostBiases = (double*)malloc(sizeof(double) * net->numOutputs[i]);
		// copying the weights and biases from gpu memory to host memory
		cudaErrorCheck(hipMemcpy(hostWeights, net->weights[i], sizeof(double) * net->numInputs[i] * net->numOutputs[i], hipMemcpyDeviceToHost));
		cudaErrorCheck(hipMemcpy(hostBiases, net->biases[i], sizeof(double) *  net->numOutputs[i], hipMemcpyDeviceToHost));

		// writing with fprint
		//writing weights first
		for (int j = 0;j < net->numOutputs[i];j++) {
			for (int z = 0;z < net->numInputs[i]; z++) {
				if (z == net->numInputs[i] - 1) {
					fprintf(toWrite, "%.15lf\n", hostWeights[(j*net->numInputs[i])+z]);
				}
				else {
					fprintf(toWrite, "%.15lf ", hostWeights[(j*net->numInputs[i])+z]);
				}
			}
		}

		//writing biases
		for (int j = 0;j < net->numOutputs[i];j++) {
			fprintf(toWrite, "%.15lf\n", hostBiases[j]);
		}

		// freeing host memory,closing file, and resetting weight file name
		free(hostWeights);
		free(hostBiases);
		fclose(toWrite);
		weightFile = string("weights.txt");
	}
}

// does the initial step of backpropogation while evaluating (sets the adjustments to be the derivative values) (we are using sigmoid, so the derivative is just output(1-output)
__global__ void backPropogateGPUInputHelper(double* weightAdjustment, double* outputs, int numInputs, int numOutputs) {
	int tidx = blockDim.x * blockIdx.x + threadIdx.x;
	int tidy = blockDim.y * blockIdx.y + threadIdx.y;
	for (int i = tidx;i < numOutputs; i += blockDim.x * gridDim.x) {
		for (int j = tidy;j < numInputs;j += blockDim.y * gridDim.y) {
			weightAdjustment[(i * numInputs) + j] = outputs[i] * (1-outputs[i]);
		}
	}
}

// evaluating the entire gpu net with cublas and some input
void evaluateGPUNet(GPUNet* toEvaluate, double* inputs, double* outputBuffer) {
	cudaErrorCheck(hipMemcpy(toEvaluate->layerInput[0], inputs, sizeof(double) * toEvaluate->numInputs[0], hipMemcpyHostToDevice));
	// going through every layer and applying cublas
	//wrapping multiplication with cublas	
	/*
	int* numInputs = (int*) malloc(sizeof(int) * toEvaluate->numLayers);
	int* numOutputs = (int*)malloc(sizeof(int) * toEvaluate->numLayers);
	cudaErrorCheck(hipMemcpy(numInputs, toEvaluate->numInputs, sizeof(int) * toEvaluate->numLayers, hipMemcpyDeviceToHost));
	cudaErrorCheck(hipMemcpy(numOutputs, toEvaluate->numOutputs, sizeof(int) * toEvaluate->numLayers, hipMemcpyDeviceToHost));
	*/

	//layer input and output to keep track of 
	double* layerOutput;

	// sizes for 2d kernels
	dim3 blockShape(16, 16);
	dim3 gridShape(32, 32);
	
	//initializing cublas handle and setting matrices
	hipblasHandle_t handle;
	hipblasStatus_t status;
	hipblasCreate(&handle);
	// looping for gpu multiplication and addition of layers
	for (int i = 0;i < toEvaluate->numLayers;i++) {
		
		int m = 1;
		int k = toEvaluate->numInputs[i];
		int n = toEvaluate->numOutputs[i];
		double identityScalar = 1.0;

		

		//allocating buffer for the input
		/*
		cudaErrorCheck(hipMalloc((&layerInput), sizeof(double) * toEvaluate->numInputs[i]));
		if (i == 0) {
			cudaErrorCheck(hipMemcpy(layerInput, inputs, sizeof(double) * toEvaluate->numOutputs[0], hipMemcpyHostToDevice));
		}
		else {
			cudaErrorCheck(hipMemcpy(layerInput, layerOutput, sizeof(double) * toEvaluate->numInputs[i], hipMemcpyDeviceToDevice));
			cudaErrorCheck(hipFree(layerOutput));
		} */

		//setting up output as a copy of biases
		cudaErrorCheck(hipMalloc(&layerOutput, sizeof(double) * toEvaluate->numOutputs[i]));
		cudaErrorCheck(hipMemcpy(layerOutput, toEvaluate->biases[i], sizeof(double) * toEvaluate->numOutputs[i], hipMemcpyDeviceToDevice));
		
		//calling cublas matrix multiply and adding biases vector (this does deviceWeights*deviceInputs + biasVector) and stores the result in the layerOutput vector

		status = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &identityScalar, toEvaluate->layerInput[i], m, toEvaluate->weights[i], k, &identityScalar, layerOutput, m);

		if (status != HIPBLAS_STATUS_SUCCESS) {
			printf("error with cublas matrix multiplication\n");
		}

		// applying sigmoid to the output
		
		sigmoidMatrix <<<20,512 >>> (layerOutput, toEvaluate->numOutputs[i]);
		cudaErrorCheck(hipGetLastError());

		//double* sigmoidedCheck = (double*)malloc(sizeof(double) * toEvaluate->numInputs[i] * toEvaluate->numOutputs[i]);
		//cudaErrorCheck(hipMemcpy(sigmoidedCheck,layerOutput,sizeof(double)*toEvaluate->numOutputs[i]))
		

		/*
		double* toSigmoid = (double*)malloc(sizeof(double) * toEvaluate->numInputs[i] * toEvaluate->numOutputs[i]);
		hipMemcpy(toSigmoid, layerOutput, sizeof(double) * toEvaluate->numInputs[i] * toEvaluate->numOutputs[i], hipMemcpyDeviceToHost);
		cpuSigmoidMatrix(toSigmoid, toEvaluate->numInputs[i] * toEvaluate->numOutputs[i]);
		hipMemcpy(layerOutput, toSigmoid, sizeof(double) * toEvaluate->numInputs[i] * toEvaluate->numOutputs[i], hipMemcpyHostToDevice);
		free(toSigmoid);
		*/

		//freeing device memory
		if (i == toEvaluate->numLayers - 1) {
			//copying output to final buffer
			cudaErrorCheck(hipMemcpy(outputBuffer, layerOutput, sizeof(double) * toEvaluate->numOutputs[i], hipMemcpyDeviceToHost));
		}
		else {
			//copying output to next layers input
			cudaErrorCheck(hipMemcpy(toEvaluate->layerInput[i + 1], layerOutput, sizeof(double) * toEvaluate->numOutputs[i], hipMemcpyDeviceToDevice));
		}
		//doing the propogation helper step
		backPropogateGPUInputHelper <<<gridShape, blockShape>>> (toEvaluate->weightAdjustments[i], layerOutput, toEvaluate->numInputs[i], toEvaluate->numOutputs[i]);
		cudaErrorCheck(hipGetLastError());
		cudaErrorCheck(hipFree(layerOutput));

	}
	//destroying handle
	hipblasDestroy(handle);
}



// calculates the adjustments based on the derivatives and sets up derivatives for next layer of backpropogation
__global__ void weightAdjust(double* weightAdjustments,double* biases, double* weights, double* derivatives, double* nextDerivatives, int numInputs, int numOutputs, double learningRate) {
	int tidx = blockDim.x * blockIdx.x + threadIdx.x;
	int tidy = blockDim.y * blockIdx.y + threadIdx.y;
	for (int i = tidx; i < numOutputs;i += gridDim.x * blockDim.x) {
		for (int j = tidy;j < numInputs;j += gridDim.y * blockDim.y) {
			weightAdjustments[(i * numInputs) + j] *= derivatives[i];
			atomicAdd(&(nextDerivatives[j]),weightAdjustments[(i * numInputs) + j] * weights[(i*numInputs)+j] );
		}
		if (tidy == 0) {
			biases[i] -= learningRate * weightAdjustments[(i * numInputs)];
		}
	}
}

//actually adjusts the weights and biases
__global__ void finalizeWeightAdjust(double* weights, double* weightAdjustments, double* inputs, int numInputs, int numOutputs,double learningRate) {
	int tidx = blockIdx.x * blockDim.x + threadIdx.x;
	int tidy = blockIdx.y * blockDim.y + threadIdx.y;
	for (int i = tidx;i < numOutputs;i += gridDim.x * blockDim.x) {
		for (int j = tidy;j < numInputs; j+= gridDim.y*blockDim.y) {
			weights[(i*numInputs) + j] -= learningRate * weightAdjustments[(i*numInputs)+j] * inputs[j];
		}
	}
}

// backpropogation of the entire net given the output values and the actual values
void backPropogateGPUNet(GPUNet* toBackProp, double* outputBuffer, double* actualRed, double* actualGreen, double* actualBlue, double learningRate) {
	// going through each layer and setting weight adjustments	
	// then performing the adjustments
	double* derivatives = (double*)malloc(sizeof(double) * outputSize);
	double* nextDerivatives;
	//setting the initial partial derivatives
	for (int i = 0;i < outputSize;i+=3) {
		derivatives[i]=(-(actualRed[i] - outputBuffer[i]));
		derivatives[i+1]=(-(actualGreen[i] - outputBuffer[i+1]));
		derivatives[i+2]=(-(actualBlue[i] - outputBuffer[i+2]));
	}

	dim3 dimBlock(16, 16);
	dim3 dimGrid;
	dimGrid.x = 32;
	dimGrid.y = 32;

	//copying derivatives to deviceDerivatives pointer
	double* deviceDerivatives;
	hipMalloc(&deviceDerivatives, sizeof(double) * outputSize);
	cudaErrorCheck(hipMemcpy(deviceDerivatives, derivatives, sizeof(double) * outputSize, hipMemcpyHostToDevice));

	for (int z = toBackProp->numLayers-1;z >= 0;z--) {
		cudaErrorCheck(hipMalloc(&nextDerivatives, sizeof(double) * toBackProp->numInputs[z]));
		weightAdjust<<<dimGrid,dimBlock>>>(toBackProp->weightAdjustments[z], toBackProp->biases[z], toBackProp->weights[z], deviceDerivatives,nextDerivatives, toBackProp->numInputs[z], toBackProp->numOutputs[z], learningRate);
		//freeing memory and setting up for next iteration
		cudaErrorCheck(hipFree(deviceDerivatives));
		deviceDerivatives = nextDerivatives;
	}

	cudaErrorCheck(hipFree(deviceDerivatives));

	//actually doing the weight adjustments
	for (int z = 0;z < toBackProp->numLayers;z++) {	
		finalizeWeightAdjust<<<dimGrid,dimBlock>>>(toBackProp->weights[z], toBackProp->weightAdjustments[z], toBackProp->layerInput[z],toBackProp->numInputs[z] ,toBackProp->numOutputs[z], learningRate);
	}

	// freeing host memory
	free(derivatives);
	
}

// given an image, we will run the net on it and output the result image
void outputFromGPUNet(char* imageName) {
	//will divide the image into squares and output it
}


// gpu trains the neural net on a random image from the dataset given a learning rate
void trainFromDataSet(double learningRate) {
	// loading the net
	GPUNet* toTrain = loadGPUNet();
	
	size_t freeMem;
	size_t totalMem;
	hipMemGetInfo(&freeMem, &totalMem);
	cout << "total memory of gpu: " << totalMem << "\n";
	cout << "total free memory of gpu after loading net: " << freeMem << "\n";
	

	
	while (true) {
		
		// pick a random image from the training dataset
		CImg<int> randomImage = getRandomTrainingImage();
		// converting image to black and white
		int* bwBuffer = (int*)malloc(sizeof(int) * randomImage.height() * randomImage.width());
		makeImageBlackAndWhiteWrapper(randomImage.data(), randomImage.data() + (randomImage.height() * randomImage.width()), randomImage.data() + (2 * randomImage.height() * randomImage.width()), bwBuffer, randomImage.height(), randomImage.width());
		// crop parts to fit neural net input size
		// we will crop into perfect squares and then combine them to get the final image (we do not need to combine them for training though)
		int trainCount = 0;
		//R,G,B training errors
		double currTrainingError[3]{};
		while (trainCount != epochNum) {
			for (int i = 0;i < randomImage.height();i += squareSide) {
				for (int j = 0;j < randomImage.width();j += squareSide) {
					// getting the square for both the bw image and color image
					int* bwSquare = (int*)malloc(sizeof(int) * squareSide * squareSide);
					int* redSquare = (int*)malloc(sizeof(int) * squareSide * squareSide);
					int* greenSquare = (int*)malloc(sizeof(int) * squareSide * squareSide);
					int* blueSquare = (int*)malloc(sizeof(int) * squareSide * squareSide);
					getSquareWrapper(bwBuffer, bwSquare, squareSide, randomImage.height(), randomImage.width(), i, j);
					getSquareWrapper(randomImage.data(), redSquare, squareSide, randomImage.height(), randomImage.width(), i, j);
					getSquareWrapper(randomImage.data() + (randomImage.height() * randomImage.width()), greenSquare, squareSide, randomImage.height(), randomImage.width(), i, j);
					getSquareWrapper(randomImage.data() + (2 * randomImage.height() * randomImage.width()), blueSquare, squareSide, randomImage.height(), randomImage.width(), i, j);

					// scale pixels by 255 for both bw image and color image
					double* scaledBWSquare = (double*)malloc(sizeof(double) * squareSide * squareSide);
					pixelScaleWrapper(bwSquare, scaledBWSquare, squareSide, squareSide);
					free(bwSquare);
					double* scaledRedSquare = (double*)malloc(sizeof(double) * squareSide * squareSide);
					pixelScaleWrapper(redSquare, scaledRedSquare, squareSide, squareSide);
					free(redSquare);
					double* scaledGreenSquare = (double*)malloc(sizeof(double) * squareSide * squareSide);
					pixelScaleWrapper(greenSquare, scaledGreenSquare, squareSide, squareSide);
					free(greenSquare);
					double* scaledBlueSquare = (double*)malloc(sizeof(double) * squareSide * squareSide);
					pixelScaleWrapper(blueSquare, scaledBlueSquare, squareSide, squareSide);
					free(blueSquare);

					// evaluate net for each part of the image
					// we will have outputSize number of outputs, and we will train the net so that the output[0] is first pixels R, output[1],output[2] represent first pixels G and B value
					double* outputBuffer = (double*)malloc(sizeof(double) * outputSize);
					
					/*
					hipMemGetInfo(&freeMem, &totalMem);
					cout << "total free memory of gpu before evaluating net: " << freeMem << "\n";
					*/

					evaluateGPUNet(toTrain, scaledBWSquare, outputBuffer);

					/*
					hipMemGetInfo(&freeMem, &totalMem);
					cout << "total free memory of gpu after evaluating net: " << freeMem << "\n";
					*/

					// printing out the current training error
					for (int i = 0;i < outputSize;i += 3) {
						currTrainingError[0] += pow(scaledRedSquare[i] - outputBuffer[i], 2);
						currTrainingError[1] += pow(scaledGreenSquare[i] - outputBuffer[i + 1], 2);
						currTrainingError[2] += pow(scaledBlueSquare[i] - outputBuffer[i + 2], 2);
					}
					/*
					cout << "red error: " << currTrainingError[0] << "\n";
					cout << "green error: " << currTrainingError[1] << "\n";
					cout << "blue error: " << currTrainingError[2] << "\n";
					*/

					//incrementTrainingErrorGPU()
					/*
					hipMemGetInfo(&freeMem, &totalMem);
					cout << "total free memory of gpu before backpropogation: " << freeMem << "\n";
					*/

					backPropogateGPUNet(toTrain, outputBuffer, scaledRedSquare, scaledGreenSquare, scaledBlueSquare, learningRate);
					
					/*
					hipMemGetInfo(&freeMem, &totalMem);
					cout << "total free memory of gpu after backpropogation: " << freeMem << "\n";
					*/

					// freeing memory we no longer need
					free(scaledBWSquare);
					free(scaledRedSquare);
					free(scaledGreenSquare);
					free(scaledBlueSquare);
					free(outputBuffer);
				}
			}
			trainCount++;
			cout << "FINISHED EPOCH: " << trainCount << "\n";
		}

		//freeing allocated memory
		free(bwBuffer);
		// then we will perform a test of error on a random test data image
		// writing weights back to filesystem now that the epochLimit was reached
		cout << "writing updated weights to filesystem\n";
		writeGPUNet(toTrain);
	}	
}

// gpu tests the neural net error on a specific image
void testImage(char* imageName) {
	//grabbing image

	//cropping parts to fit neural net input size

	//scale pixels by 255

	//evaluate net for each part

	//calculate error
}

